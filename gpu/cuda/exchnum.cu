#include "hip/hip_runtime.h"
/* -*- mode: c -*- */
#include <cassert>
#include <iostream>
#include <fstream>
#include <map>
#include <string>
#include "../common.h"
#include "../init.h"
#include "cuda_extra.h"
#include "../matrix.h"
#include "gpu_variables.h"
#include "../timer.h"
#include "double.h"
#include "../partition.h"

/** KERNELS **/
#include "functions.h"
#include "weight.h"

#if CPU_KERNELS
#include "../exchnum.cpp"
#else
#include "pot.h"
#include "energy.h"
#include "rmm.h"
#include "energy_derivs.h"
#include "force.h"
#endif

using namespace G2G;
using namespace std;

#define COMPUTE_RMM 					0
#define COMPUTE_ENERGY_ONLY		1
#define COMPUTE_ENERGY_FORCE	2
#define COMPUTE_FORCE_ONLY		3


/*******************************
 * Cube Functions
 *******************************/
void gpu_compute_group_functions(void)
{
	cout << "<===== computing functions ========>" << endl;
	CudaMatrixFloat4 points_position_gpu;
	CudaMatrixFloat2 factor_ac_gpu;
	CudaMatrixUInt nuc_gpu;
	CudaMatrixUInt contractions_gpu;
	
	Timer t1;
	t1.sync();
	t1.start();
	
	for (list<PointGroup>::iterator it = final_partition.begin(); it != final_partition.end(); ++it) {
		PointGroup& group = *it;
		/** Load points from group **/
		{
			HostMatrixFloat4 points_position_cpu(group.number_of_points, 1);
						
			uint i = 0;		
			for (list<Point>::const_iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
				points_position_cpu.get(i) = make_float4(p->position.x, p->position.y, p->position.z, 0);
			}
			points_position_gpu = points_position_cpu;
		}
		
		/* Load group functions */
		uint group_m = group.s_functions + group.p_functions * 3 + group.d_functions * 6;
		uint4 group_functions = make_uint4(group.s_functions, group.p_functions, group.d_functions, group_m);
		{
			HostMatrixFloat2 factor_ac_cpu(COALESCED_DIMENSION(group_m), MAX_CONTRACTIONS);
			HostMatrixUInt nuc_cpu(group_m, 1), contractions_cpu(group_m, 1);
			
			uint i = 0;
      uint ii = 0;
			for (set<uint>::const_iterator func = group.functions.begin(); func != group.functions.end(); ++func, ++i) {
        uint inc;
        if (i < group.s_functions) inc = 1;
        else if (i < group.s_functions + group.p_functions) inc = 3;
        else inc = 6;

        uint this_nuc = fortran_vars.nucleii.get(*func) - 1;
        uint this_cont = fortran_vars.contractions.get(*func);

        for (uint j = 0; j < inc; j++) {
          nuc_cpu.get(ii) = this_nuc;
          contractions_cpu.get(ii) = this_cont;
  				for (unsigned int k = 0; k < this_cont; k++)
            factor_ac_cpu.get(ii, k) = make_float2(fortran_vars.a_values.get(*func, k), fortran_vars.c_values.get(*func, k));
          ii++;
        }				
			}

			factor_ac_gpu = factor_ac_cpu;
			nuc_gpu = nuc_cpu;
			contractions_gpu = contractions_cpu;
		}
		
		/** Compute Functions **/
    group.function_values.resize(COALESCED_DIMENSION(group.number_of_points), group_functions.w);
    if (fortran_vars.do_forces) group.gradient_values.resize(COALESCED_DIMENSION(group.number_of_points), group_functions.w);
		
		dim3 threads(group.number_of_points);
		dim3 threadBlock(FUNCTIONS_BLOCK_SIZE);
		dim3 threadGrid = divUp(threads, threadBlock);		

		//cout << "points: " << threads.x << " " << threadGrid.x << " " << threadBlock.x << endl;
		
		if (fortran_vars.do_forces)
			gpu_compute_functions<true><<<threadGrid, threadBlock>>>(points_position_gpu.data, group.number_of_points, contractions_gpu.data, factor_ac_gpu.data, nuc_gpu.data, group.function_values.data, group.gradient_values.data, group_functions);
		else
			gpu_compute_functions<false><<<threadGrid, threadBlock>>>(points_position_gpu.data, group.number_of_points, contractions_gpu.data, factor_ac_gpu.data, nuc_gpu.data, group.function_values.data, group.gradient_values.data, group_functions);

		cudaAssertNoError("compute_functions");
	}	
	
	t1.sync();
	t1.stop();
	cout << "TIMER: funcs: " << t1 << endl;

  //cudaPrintMemoryInfo();
}

/*******************************
 * Cube Weights
 *******************************/

void gpu_compute_group_weights(PointGroup& group)
{
  //cout << "group" << endl;
  CudaMatrixFloat4 point_positions_gpu;
  CudaMatrixFloat4 atom_position_rm_gpu;
  CudaMatrixUInt nucleii_gpu;
  {
    HostMatrixFloat4 points_positions_cpu(group.number_of_points, 1);
		uint i = 0;
		for (list<Point>::const_iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
			points_positions_cpu.get(i) = make_float4(p->position.x, p->position.y, p->position.z, p->atom);
		}
    point_positions_gpu = points_positions_cpu;

    HostMatrixFloat4 atom_position_rm_cpu(fortran_vars.atoms, 1);
    for (uint i = 0; i < fortran_vars.atoms; i++) {
      double3 atom_pos = fortran_vars.atom_positions.get(i);
      atom_position_rm_cpu.get(i) = make_float4(atom_pos.x, atom_pos.y, atom_pos.z, fortran_vars.rm.get(i));
    }
    atom_position_rm_gpu = atom_position_rm_cpu;
    
    HostMatrixUInt nucleii_cpu(group.nucleii.size(), 1);
    i = 0;
    for (set<uint>::iterator it = group.nucleii.begin(); it != group.nucleii.end(); ++it, i++) {
      nucleii_cpu.get(i) = *it;
    }
    nucleii_gpu = nucleii_cpu;
	}

  CudaMatrixFloat weights_gpu(group.number_of_points);
  dim3 threads(group.number_of_points);
  dim3 blockSize(WEIGHT_BLOCK_SIZE);
  dim3 gridSize = divUp(threads, blockSize);
  gpu_compute_weights<<<gridSize,blockSize>>>(group.number_of_points, point_positions_gpu.data, atom_position_rm_gpu.data,
                                              weights_gpu.data, nucleii_gpu.data, group.nucleii.size());
  cudaAssertNoError("compute_weights");

  #if REMOVE_ZEROS
  std::list<Point> nonzero_points;
  uint nonzero_number_of_points = 0;
  #endif

  uint ceros = 0;

  HostMatrixFloat weights_cpu(weights_gpu);
  uint i = 0;
  for (list<Point>::iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
    p->weight *= weights_cpu.get(i);

    if (p->weight == 0.0) {
      ceros++;
    }
    #if REMOVE_ZEROS
    else {
      nonzero_points.push_back(*p);
      nonzero_number_of_points++;
    }
    #endif
  }

  //cout << "ceros: " << ceros << "/" << group.number_of_points << " (" << (ceros / (double)group.number_of_points) * 100 << "%)" << endl;
  
  #if REMOVE_ZEROS
  group.points = nonzero_points;
  group.number_of_points = nonzero_number_of_points;
  #endif
}

/********************************
 * Solve Cubes
 ********************************/
extern "C" void g2g_solve_groups_(uint& computation_type, double* fort_energy_ptr, double* fort_forces_ptr)
{
	cout << "<================ calculo de: [";
	switch(computation_type) {
		case COMPUTE_ENERGY_ONLY: cout << "energia"; break;
		case COMPUTE_RMM: cout << "rmm"; break;
		case COMPUTE_FORCE_ONLY: cout << "fuerzas"; break;
		case COMPUTE_ENERGY_FORCE: cout << "energia+fuerzas"; break;
	}
	cout << "] ==========>" << endl;
	
	Timer t_total;
	t_total.sync();
	t_total.start();
		
	/*** Computo sobre cada cubo ****/
	CudaMatrixFloat point_weights_gpu;
	CudaMatrixFloat rdm_gpu, rdmt_gpu;
  CudaMatrixUInt nuc_gpu;
  CudaMatrixFloat2 factor_ac_gpu;
	CudaMatrixUInt2 nuc_contractions_gpu;

  Timer t_density, t_rmm, t_forces;
  Timer t_cpu;

  uint max_used_memory = 0;

	FortranMatrix<double> fort_forces(fort_forces_ptr, fortran_vars.atoms, 3, FORTRAN_MAX_ATOMS);
	
	double total_energy = 0.0;
		
	for (list<PointGroup>::const_iterator it = final_partition.begin(); it != final_partition.end(); ++it) {
		const PointGroup& group = *it;
				
		/** Load points from group **/
    HostMatrixFloat point_weights_cpu(group.number_of_points, 1);

		uint i = 0;		
		for (list<Point>::const_iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
			point_weights_cpu.get(i) = p->weight;
		}
    #if !CPU_KERNELS
		point_weights_gpu = point_weights_cpu;
    #endif
		
		/** Load functions from group **/
		uint group_m = group.s_functions + group.p_functions * 3 + group.d_functions * 6;
		uint4 group_functions = make_uint4(group.s_functions, group.p_functions, group.d_functions, group_m);
		{
			HostMatrixFloat2 factor_ac_cpu(COALESCED_DIMENSION(group_m), MAX_CONTRACTIONS);
			HostMatrixUInt2 nuc_contractions_cpu(group_m, 1);
      HostMatrixUInt nuc_cpu(group_m, 1);

			uint i = 0;
      uint ii = 0;
			for (set<uint>::const_iterator func = group.functions.begin(); func != group.functions.end(); ++func, ++i) {
        uint inc;
        if (i < group.s_functions) inc = 1;
        else if (i < group.s_functions + group.p_functions) inc = 3;
        else inc = 6;

        uint this_nuc = fortran_vars.nucleii.get(*func) - 1;
        uint this_cont = fortran_vars.contractions.get(*func);

        for (uint j = 0; j < inc; j++) {
          nuc_cpu.get(ii) = this_nuc;

          nuc_contractions_cpu.get(ii) = make_uint2(this_nuc, this_cont);
  				for (unsigned int k = 0; k < this_cont; k++)
            factor_ac_cpu.get(ii, k) = make_float2(fortran_vars.a_values.get(*func, k), fortran_vars.c_values.get(*func, k));
  				for (unsigned int k = this_cont; k < MAX_CONTRACTIONS; k++)
            factor_ac_cpu.get(ii, k) = make_float2(0.0f,0.0f);

          ii++;
        }
			}

			factor_ac_gpu = factor_ac_cpu;
			nuc_contractions_gpu = nuc_contractions_cpu;
      nuc_gpu = nuc_cpu;
		}

		/* load RDM */
    HostMatrixFloat rdm_cpu(COALESCED_DIMENSION(group_m), fortran_vars.nco);
    HostMatrixFloat rdmt_cpu(COALESCED_DIMENSION(fortran_vars.nco), group_m);

    for (unsigned int i = 0; i < fortran_vars.nco; i++) {
      uint j = 0;
      for (set<uint>::const_iterator func = group.functions.begin(); func != group.functions.end(); ++func) {
        if (*func < fortran_vars.s_funcs) {
          rdm_cpu.get(j, i) = fortran_vars.rmm_input.get(*func, i);
          rdmt_cpu.get(i, j) = rdm_cpu.get(j, i);
          j++;
        }
        else if (*func < (fortran_vars.s_funcs + fortran_vars.p_funcs * 3)) {
          for (uint k = 0; k < 3; k++, j++) {
            rdm_cpu.get(j, i) = fortran_vars.rmm_input.get(*func + k, i);
            rdmt_cpu.get(i, j) = rdm_cpu.get(j, i);
          }
        }
        else {
          for (uint k = 0; k < 6; k++, j++) {
            rdm_cpu.get(j, i) = fortran_vars.rmm_input.get(*func + k, i);
            rdmt_cpu.get(i, j) = rdm_cpu.get(j, i);
          }
        }
      }
    }
    #if !CPU_KERNELS
    rdm_gpu = rdm_cpu;
    rdmt_gpu = rdmt_cpu;
    #endif

		dim3 threads(group.number_of_points);
		dim3 threadBlock, threadGrid;
		threadBlock = dim3(DENSITY_BLOCK_SIZE);
		threadGrid = divUp(threads, threadBlock);

		/* compute energy */
		if (computation_type == COMPUTE_ENERGY_ONLY) {
      #if CPU_KERNELS
      HostMatrixFloat energy_cpu(group.number_of_points);
      HostMatrixFloat function_values_cpu(group.function_values.height, group.function_values.width);
      function_values_cpu.copy_transpose(group.function_values);

      t_cpu.start_and_sync();
      cpu_compute_density_forces<true, false>(energy_cpu.data, point_weights_cpu.data, group.number_of_points, rdmt_cpu.data,
        NULL, function_values_cpu.data, NULL, NULL, NULL, 0, group_m, t_density, t_rmm);
      t_cpu.pause_and_sync();
      #else
      t_density.start_and_sync();
			CudaMatrixFloat energy_gpu(group.number_of_points);
			gpu_compute_density<true, false><<<threadGrid, threadBlock>>>(energy_gpu.data, NULL, point_weights_gpu.data, group.number_of_points,
                                                                    rdmt_gpu.data, group.function_values.data, group_m, NULL);
			cudaAssertNoError("compute_density");
      t_density.pause_and_sync();
      HostMatrixFloat energy_cpu(energy_gpu);
      #endif
			
			for (uint i = 0; i < group.number_of_points; i++) { total_energy += energy_cpu.get(i); }

      uint free_memory, total_memory;
      cudaGetMemoryInfo(free_memory, total_memory);
      max_used_memory = max(max_used_memory, total_memory - free_memory);
		}
		/* compute necessary factor **/
		else if (computation_type == COMPUTE_RMM) {
      #if CPU_KERNELS

      HostMatrixFloat function_values_cpu(group.function_values.height, group.function_values.width);
      function_values_cpu.copy_transpose(group.function_values);
      HostMatrixFloat rmm_output_cpu(COALESCED_DIMENSION(group_m), group_m);
      t_cpu.start_and_sync();
      cpu_compute_density_forces<false, false>(NULL, point_weights_cpu.data, group.number_of_points, rdmt_cpu.data,
        rmm_output_cpu.data, function_values_cpu.data, NULL, NULL, NULL, 0, group_m, t_density, t_rmm);
      t_cpu.pause_and_sync();
      
      #else

			CudaMatrixFloat rmm_factor_gpu(group.number_of_points);
      t_density.start_and_sync();
			gpu_compute_density<false, false><<<threadGrid, threadBlock>>>(NULL, rmm_factor_gpu.data, point_weights_gpu.data, group.number_of_points,
                                                                     rdmt_gpu.data, group.function_values.data, group_m, NULL);
			cudaAssertNoError("compute_density");
      t_density.pause_and_sync();

			/*** Compute RMM update ***/
			threads = dim3(group_m, group_m);
			threadBlock = dim3(RMM_BLOCK_SIZE_XY, RMM_BLOCK_SIZE_XY);
			threadGrid = divUp(threads, threadBlock);

      CudaMatrixFloat rmm_output_gpu(COALESCED_DIMENSION(group_m), group_m);
      t_rmm.start_and_sync();
			gpu_update_rmm<<<threadGrid, threadBlock>>>(rmm_factor_gpu.data, group.number_of_points, rmm_output_gpu.data, group.function_values.data, group_m);
			cudaAssertNoError("update_rmm");
      t_rmm.pause_and_sync();

			HostMatrixFloat rmm_output_cpu(rmm_output_gpu);
      #endif

      /*** Contribute this RMM to the total RMM ***/
      uint small_fi = 0;

			for (set<uint>::iterator it_fi = group.functions.begin(); it_fi != group.functions.end(); ++it_fi) {
				uint fi_advance;
				if (*it_fi < fortran_vars.s_funcs) fi_advance = 1;
				else if (*it_fi < fortran_vars.s_funcs + fortran_vars.p_funcs * 3) fi_advance = 3;
				else fi_advance = 6;
				
				for (uint i = 0; i < fi_advance; i++) {

          uint small_fj = 0;
					for (set<uint>::iterator it_fj = group.functions.begin(); it_fj != group.functions.end(); ++it_fj) {
						uint fj_advance;
						if (*it_fj < fortran_vars.s_funcs) fj_advance = 1;
						else if (*it_fj < fortran_vars.s_funcs + fortran_vars.p_funcs * 3) fj_advance = 3;
						else fj_advance = 6;
					
						for (uint j = 0; j < fj_advance; j++) {
							uint fi = *it_fi + i; uint fj = *it_fj + j;
							if (fi > fj) continue;
							uint big_index = (fi * fortran_vars.m - (fi * (fi - 1)) / 2) + (fj - fi);
              fortran_vars.rmm_output.get(big_index) += rmm_output_cpu.get(small_fi, small_fj + small_fi);
              small_fj++;
						}					
					}
          small_fi++;
				}
			}

      uint free_memory, total_memory;
      cudaGetMemoryInfo(free_memory, total_memory);
      max_used_memory = max(max_used_memory, total_memory - free_memory);
		}
    #if 0
		/* compute forces */
		else {
      map<uint, uint> nuc_mapping;
			HostMatrixUInt nuc_cpu(group_m, 1);
			uint i = 0;
      uint small_atom_idx = 0;

      for (set<uint>::iterator func = group.functions.begin(); func != group.functions.end(); ++func) {
        uint f_advance;
        if (*func < fortran_vars.s_funcs) f_advance = 1;
        else if (*func < fortran_vars.s_funcs + fortran_vars.p_funcs * 3) f_advance = 3;
        else f_advance = 6;

        for (uint j = 0; j < f_advance; j++, i++) {
          uint big_atom_idx = fortran_vars.nucleii.get(*func) - 1;
          if (nuc_mapping.find(big_atom_idx) == nuc_mapping.end()) {
            nuc_mapping[big_atom_idx] = small_atom_idx;
            small_atom_idx++;
          }
          nuc_cpu.get(i) = nuc_mapping[big_atom_idx];
        }
      }
      #if !CPU_KERNELS
			nuc_gpu = nuc_cpu;
      #endif

      #if CPU_KERNELS
      HostMatrixFloat energy_cpu;;
      HostMatrixFloat4 forces_cpu(group.nucleii.size());

      if (computation_type == COMPUTE_ENERGY_FORCE) {
        energy_cpu.resize(group.number_of_points);
        HostMatrixFloat function_values_cpu(group.function_values.height, group.function_values.width);
        function_values_cpu.copy_transpose(group.function_values);
        HostMatrixFloat4 gradient_values_cpu(group.gradient_values.height, group.gradient_values.width);
        gradient_values_cpu.copy_transpose(group.gradient_values);

        t_cpu.start_and_sync();
        cpu_compute_density_forces<true, true>(energy_cpu.data, point_weights_cpu.data, group.number_of_points, rdm_cpu.data, NULL,
          function_values_cpu.data, gradient_values_cpu.data, forces_cpu.data, nuc_cpu.data, group.nucleii.size(), group_m, t_density, t_rmm);
        t_cpu.pause_and_sync();

        for (uint i = 0; i < group.number_of_points; i++) { total_energy += energy_cpu.get(i); }
      }
      else {
        HostMatrixFloat function_values_cpu(group.function_values.height, group.function_values.width);
        function_values_cpu.copy_transpose(group.function_values);
        HostMatrixFloat4 gradient_values_cpu(group.gradient_values.height, group.gradient_values.width);
        gradient_values_cpu.copy_transpose(group.gradient_values);
        
        t_cpu.start_and_sync();
        cpu_compute_density_forces<false,true>(NULL, point_weights_cpu.data, group.number_of_points, rdm_cpu.data, NULL,
          function_values_cpu.data, gradient_values_cpu.data, forces_cpu.data, nuc_cpu.data, group.nucleii.size(), group_m, t_density, t_rmm);
        t_cpu.pause_and_sync();
      }
      #else

			CudaMatrixFloat force_factor_gpu(group.number_of_points);
			CudaMatrixFloat energy_gpu;
      CudaMatrixFloat w_gpu(COALESCED_DIMENSION(group.number_of_points), fortran_vars.nco);

			/* energy may be needed at this step */
			CudaMatrixFloat4 density_deriv(COALESCED_DIMENSION(group.number_of_points), group.nucleii.size());
			if (computation_type == COMPUTE_ENERGY_FORCE) {
				energy_gpu.resize(group.number_of_points);
        t_density.start_and_sync();

				gpu_compute_density<true, true><<<threadGrid, threadBlock>>>(energy_gpu.data, force_factor_gpu.data, point_weights_gpu.data, group.number_of_points,
                                                               rdm_gpu.data, group.function_values.data, group_m, w_gpu.data);
        cudaAssertNoError("compute_density");
        t_density.pause_and_sync();

				HostMatrixFloat energy_cpu(energy_gpu);
				for (uint i = 0; i < group.number_of_points; i++) { total_energy += energy_cpu.get(i); }
			}
			else {
        t_density.start_and_sync();
				gpu_compute_density<false, true><<<threadGrid, threadBlock>>>(energy_gpu.data, force_factor_gpu.data, point_weights_gpu.data, group.number_of_points,
                                                                rdm_gpu.data, group.function_values.data, group_m, w_gpu.data);
        t_density.pause_and_sync();
        cudaAssertNoError("compute_density");
      }

      threadBlock = dim3(DENSITY_DERIV_BLOCK_SIZE);
      threadGrid = divUp(threads, threadBlock);

      t_density.start_and_sync();
			gpu_compute_density_derivs<<<threadGrid, threadBlock>>>(group.number_of_points, rdmt_gpu.data, group.gradient_values.data, density_deriv.data, nuc_gpu.data,
                                                              group.nucleii.size(), group_m, w_gpu.data);
      t_density.pause_and_sync();
      cudaAssertNoError("compute_density_deriv");

			threads = dim3(group.nucleii.size());
			threadBlock = dim3(FORCE_BLOCK_SIZE);
			threadGrid = divUp(threads, threadBlock);

			CudaMatrixFloat4 forces_gpu(group.nucleii.size());
      t_forces.start_and_sync();
			gpu_compute_forces<<<threadGrid, threadBlock>>>(group.number_of_points, force_factor_gpu.data, density_deriv.data, forces_gpu.data, group.nucleii.size());
      t_forces.pause_and_sync();
			cudaAssertNoError("gpu_compute_forces");
      HostMatrixFloat4 forces_cpu(forces_gpu);
      #endif

			for (map<uint, uint>::iterator nuc_it = nuc_mapping.begin(); nuc_it != nuc_mapping.end(); ++nuc_it) {
				float4 atom_force = forces_cpu.get(nuc_it->second);
        //cout << "atom force: " << atom_force.x << " " << atom_force.y << " " << atom_force.z << endl;
				fort_forces.get(nuc_it->first, 0) += atom_force.x;
				fort_forces.get(nuc_it->first, 1) += atom_force.y;
				fort_forces.get(nuc_it->first, 2) += atom_force.z;
      }
		}
    #endif
	}
		
	/** pass results to fortran */
	if (computation_type == COMPUTE_ENERGY_ONLY || computation_type == COMPUTE_ENERGY_FORCE) {
		cout << "total energy: " << total_energy << endl;
		*fort_energy_ptr = total_energy;
	}
	t_total.stop_and_sync();

	cout << "TIMER: gpu_solve_cubes " << t_total << endl;
  cout << "TIMER: density/energy " << t_density << endl;
  cout << "TIMER: forces " << t_forces << endl;
  cout << "TIMER: rmm: " << t_rmm << endl;
  #if CPU_KERNELS
  cout << "TIMER: cpu: " << t_cpu << endl;
  #endif

  uint free_memory, total_memory;
  cudaGetMemoryInfo(free_memory, total_memory);
  cout << "Maximum used memory: " << (double)max_used_memory / (1024 * 1024) << "MB (" << ((double)max_used_memory / total_memory) * 100.0 << "%)" << endl;

}
