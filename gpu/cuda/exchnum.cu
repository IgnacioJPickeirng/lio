#include "hip/hip_runtime.h"
/* -*- mode: c -*- */
#include <cstdio>
#include "cuda_extra.h"
#include "../matrix.h"
#include "accum.h"
#include "exchnum.h"
#include "exchnum_constants.h"
#include "../timer.h"

#include <cassert>

using namespace G2G;
using namespace std;

/**
 * TODO: revisar distance / distance2 cuando sea necesario
 */

template <const uint* const curr_layers, uint grid_n>
__global__ void calc_new_rmm(const float3* atom_positions, const uint* types, const float3* point_positions,
														 const float* wang, const uint atoms_n, uint nco, uint3 num_funcs,
														 const uint* nuc, const uint* contractions, bool normalize, const float* factor_a, const float* factor_c,
														 const float* rmm, float* rmm_output, float* factors);


__device__ void calc_function(const uint3& num_funcs, const uint* nuc, const uint* contractions, const float3& point_position,
															const float3* atom_positions, const float* factor_a, const float* factor_c, uint big_func_index,
															bool normalize, float& func_value);

/***************************************** ENERGY KERNEL ******************************************/
#include "energy.h"

/*************************************** ACTUALIZACION DE RMM *************************************/
#include "rmm.h"


/**
 * Fortran interface
 */

/** TODO: ----- ARREGLAR ESTO ---------- */
#define FORTRAN_MAX_ATOMS 1860
#define FORTRAN_NG 1200
#define FORTRAN_NL 10

/**
 * Parametros innecesarios: m (es sum(num_funcs))
 */
extern "C" void exchnum_gpu_(const unsigned int& norm, const unsigned int& natom, const double* r, const unsigned int* Iz, const unsigned int* Nuc,
														 const unsigned int& m, const unsigned int* ncont, const unsigned int* nshell, const double* c, const double* a,
														 double* RMM, const unsigned int& m18, const unsigned int& m5, const unsigned int& nco, double& Exc, const unsigned int& nopt,
														 const unsigned int& Iexch, const unsigned int& igrid,
														 const double* e, const double* e2, const double* e3,
														 const double* fort_wang, const double* fort_wang2, const double* fort_wang3,
														 const unsigned int& Ndens, const unsigned int& is_int3lu)
{
	printf("<======= exchnum_gpu (from %s) ============>\n", is_int3lu ? "int3lu" : "SCF");
	printf("Ndens: %i\n", Ndens);
	uint3 num_funcs = make_uint3(nshell[0], nshell[1], nshell[2]);
	uint3 num_funcs_div = num_funcs / make_uint3(1, 3, 6);
	
	uint total_funcs = sum(num_funcs);
	uint total_funcs_div = sum(num_funcs_div);
	
	uint points = EXCHNUM_SMALL_GRID_SIZE;
	switch (igrid) {
		case 0: points = EXCHNUM_SMALL_GRID_SIZE; 	break;
		case 1: points = EXCHNUM_MEDIUM_GRID_SIZE;	break;
		case 2: points = EXCHNUM_BIG_GRID_SIZE;			break;
	}
	
	#ifdef ENERGY3D
	dim3 threads(natom, MAX_LAYERS, points);
	dim3 blockSize(1, 8, 16);
	dim3 gridSize3d = divUp(threads, blockSize);
	#else
	dim3 threads(natom, points);
	dim3 blockSize(1, 128);
	dim3 gridSize3d = divUp(threads, blockSize);
	#endif
	
	HostMatrixFloat3 atom_positions(natom), point_positions(points);
	/* output_rmm size: TODO: divUp(m * (m - 1),2) */
	HostMatrixFloat factor_a(total_funcs, MAX_CONTRACTIONS), factor_c(total_funcs, MAX_CONTRACTIONS), wang(points);
	HostMatrixUInt types(natom), nuc(total_funcs_div), contractions(total_funcs_div);
	
	// REVISAR: nuc: imagen y dominio (especialmente por la parte de * 3 y * 6)

	printf("%i atoms\n", natom);
	for (unsigned int i = 0; i < natom; i++) {
		atom_positions.data[i] = make_float3(r[FORTRAN_MAX_ATOMS * 0 + i], r[i + FORTRAN_MAX_ATOMS * 1], r[i + FORTRAN_MAX_ATOMS * 2]);
		//printf("Pos(%i): %f %f %f, Types(%i): %i\n", i, atom_positions.data[i].x, atom_positions.data[i].y, atom_positions.data[i].z, i, Iz[i]);		
		types.data[i] = Iz[i] - 1;
	}
	
	printf("ns: %i, np: %i, nd: %i, Total_Funcs: %i\n", num_funcs.x, num_funcs.y, num_funcs.z, total_funcs);
	{
		uint inc = 1;
		uint i, j;
		for (i = 0, j = 0; i < total_funcs; i += inc, j++) {
			if (i == num_funcs.x) inc = 3;
			else if (i == num_funcs.x + num_funcs.y) inc = 6;

			//printf("i: %i, j: %i\n", i, j);
			//printf("Nuc(%i) = %i\n", i, Nuc[i] - 1);
			//printf("ncont(%i) = %i\n", i, ncont[i]);
			nuc.data[j] = Nuc[i] - 1;
			contractions.data[j] = ncont[i];
			
			for (unsigned int k = 0; k < ncont[i]; k++) {
				factor_a.data[j * MAX_CONTRACTIONS + k] = a[FORTRAN_NG * k + i];
				factor_c.data[j * MAX_CONTRACTIONS + k] = c[FORTRAN_NG * k + i];
				//printf("cont: %i, a: %f, c: %f\n", k, factor_a.data[j * MAX_CONTRACTIONS + k], factor_c.data[j * MAX_CONTRACTIONS + k]);
			}			
		}
	}
	
	HostMatrixFloat rmm;
	printf("NCO: %i, M: %i, Iexch: %i\n", nco, total_funcs, Iexch);
	assert(Iexch == 1);	
	{
		if (Ndens == 1) {
			rmm.resize(m * m);
			uint k = 0;
			for (unsigned int i = 0; i < m; i++) {
				for (unsigned int j = i; j < m; j++) {
					rmm.data[k] = RMM[k];
					//printf("rmm(%i): %.30e\n", k, RMM[m5 + k - 1]);
					k++;
				}
			}
		}
		else {
			rmm.resize(m, nco);
			uint k = m18 - 1;
			for (unsigned int i = 0; i < m; i++) {
				for (unsigned int j = 0; j < nco; j++) {
					rmm.data[i * nco + j] = RMM[k];
					//printf("rmm(%i,%i): %.30e (%i)\n", i, j, RMM[k], k);
					k++;
				}
			}
		}
	}

	const double* real_e = NULL;
	const double* real_wang = NULL;
	switch (igrid) {
		case 0: real_e = e;  real_wang = fort_wang;  	break;
		case 1: real_e = e2; real_wang = fort_wang2; 	break;
		case 2: real_e = e3; real_wang = fort_wang3; 	break;		
	}

	printf("Puntos (grilla %i):\n", igrid);	
	for (unsigned int i = 0; i < points; i++) {
		wang.data[i] = real_wang[i];
		point_positions.data[i] = make_float3(real_e[0 * points + i], real_e[1 * points + i], real_e[2 * points + i]);
		//printf("wang: %f, e: (%f,%f,%f)\n", wang.data[i], point_positions.data[i].x, point_positions.data[i].y, point_positions.data[i].z);
	}
	
	HostMatrixFloat energy(1);
	calc_energy(atom_positions, types, igrid, point_positions, energy, wang,
							Ndens, nco, num_funcs_div, nuc, contractions, norm, factor_a, factor_c, rmm, &RMM[m5-1],
							is_int3lu, threads, blockSize, gridSize3d);

	if (!is_int3lu) {
		/* update fortran variables */
		Exc = energy.data[0];
		printf("Exc: %f\n", energy.data[0]);
	}
}

/**
 * Host <-> CUDA Communication function
 */

void calc_energy(const HostMatrixFloat3& atom_positions, const HostMatrixUInt& types, uint grid_type,
								 const HostMatrixFloat3& point_positions, HostMatrixFloat& energy, const HostMatrixFloat& wang,
								 uint Ndens, uint nco, uint3 num_funcs, const HostMatrixUInt& nuc,
								 const HostMatrixUInt& contractions, bool normalize, const HostMatrixFloat& factor_a, const HostMatrixFloat& factor_c,
								 const HostMatrixFloat& rmm, double* cpu_rmm_output, bool update_rmm, const dim3& threads, const dim3& blockSize, const dim3& gridSize3d)
{
	const CudaMatrixFloat3 gpu_atom_positions(atom_positions);
	const CudaMatrixUInt gpu_types(types), gpu_nuc(nuc), gpu_contractions(contractions);
	
	#ifdef ENERGY3D
	uint gridSizeZ = gridSize3d.z;
	dim3 gridSize = dim3(gridSize3d.x, gridSize3d.y * gridSize3d.z, 1);
	#else
	uint gridSizeZ = 1;
	dim3 gridSize = gridSize3d;
	#endif
	
	uint m = num_funcs.x + num_funcs.y * 3 + num_funcs.z * 6;	
	uint small_m = sum(num_funcs);
	
	uint natoms = atom_positions.width;
	uint npoints = point_positions.width;
	
	CudaMatrixFloat gpu_energy, gpu_total_energy, gpu_wang(wang), gpu_factor_a(factor_a), gpu_factor_c(factor_c),
									gpu_rmm(rmm), gpu_functions(m *  natoms * MAX_LAYERS * npoints);
	CudaMatrixFloat3 gpu_point_positions(point_positions);
	
	printf("creando espacio para funcs output: size: %i (%i bytes) data: %i\n", gpu_functions.elements(), gpu_functions.bytes(), (bool)gpu_functions.data);	

	// optional update of RMM(M5)
	CudaMatrixFloat gpu_rmm_output;
	if (update_rmm) {
		// gpu_rmm_output.resize((m * (m + 1)) / 2);
		gpu_rmm_output.resize(m * m); // TODO: ajustar
		printf("creando espacio para rmm output: size: %i (%i bytes) data: %i\n", gpu_rmm_output.elements(), gpu_rmm_output.bytes(), (bool)gpu_rmm_output.data);
	}
	#ifndef _DEBUG
	else
	#endif
		gpu_energy.resize(natoms * MAX_LAYERS * npoints);
	
	printf("threads: %i %i %i, blockSize: %i %i %i, gridSize: %i %i %i\n", threads.x, threads.y, threads.z, blockSize.x, blockSize.y, blockSize.z, gridSize.x, gridSize.y / gridSizeZ, gridSizeZ);
	if (update_rmm) printf("GPU RMM SIZE: %i (%i bytes)\n", gpu_rmm_output.elements(), gpu_rmm_output.bytes());
	printf("energy data elements: %i data: %i\n", gpu_energy.elements(), (bool)gpu_energy.data);
	// TODO: update_rmm should be a template parameter
	const uint* curr_cpu_layers = NULL;
	
	float* factor_output = NULL;
	
	CudaMatrixFloat gpu_factor_output(natoms * MAX_LAYERS * npoints);
	factor_output = gpu_factor_output.data;

	#if 0
	dim3 rmmThreads(divUp(m, 2), m + 1);
	dim3 rmmBlockSize(8,16);
	dim3 rmmGridSize = divUp(rmmThreads, rmmBlockSize);
	#else
	/*dim3 rmmThreads(m,m);
	dim3 rmmBlockSize(1);
	dim3 rmmGridSize(m,m);*/
	dim3 rmmThreads(m, m);
	dim3 rmmBlockSize(8,16);
	dim3 rmmGridSize = divUp(rmmThreads, rmmBlockSize);
	#endif
	
	printf("rmm threads: %i %i, blockSize: %i %i, gridSize: %i %i\n", rmmThreads.x, rmmThreads.y, rmmBlockSize.x, rmmBlockSize.y, rmmGridSize.x, rmmGridSize.y);		
	
	switch(grid_type) {
		case 0:
		{
			energy_kernel<EXCHNUM_SMALL_GRID_SIZE, layers2><<< gridSize, blockSize >>>(gridSizeZ, gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_energy.data,
																																								 gpu_wang.data, gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																								 normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_functions.data,
																																								 Ndens, factor_output, update_rmm);
			
			if (update_rmm) {				
				hipError_t error = hipGetLastError();
				if (error != hipSuccess) fprintf(stderr, "=!=!=!=!=====> CUDA ERROR <=====!=!=!=!=: %s\n", hipGetErrorString(error));
				
				calc_new_rmm<layers2, EXCHNUM_SMALL_GRID_SIZE><<<rmmGridSize, rmmBlockSize>>>(gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_wang.data,
																																											gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																											normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_rmm_output.data,
																																											factor_output, gpu_functions.data);
			}

			curr_cpu_layers = cpu_layers2;
		}
		break;
		case 1:
		{
			energy_kernel<EXCHNUM_MEDIUM_GRID_SIZE, layers><<< gridSize, blockSize >>>(gridSizeZ, gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_energy.data,
																																								 gpu_wang.data, gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																								 normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_functions.data, 
																																								 Ndens, factor_output, update_rmm);
			if (update_rmm) {
				hipError_t error = hipGetLastError();
				if (error != hipSuccess) fprintf(stderr, "=!=!=!=!=====> CUDA ERROR <=====!=!=!=!=: %s\n", hipGetErrorString(error));

				calc_new_rmm<layers, EXCHNUM_MEDIUM_GRID_SIZE><<<rmmGridSize, rmmBlockSize>>>(gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_wang.data,
																																											 gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																											 normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_rmm_output.data,
																																											 factor_output, gpu_functions.data);
			}

			curr_cpu_layers = cpu_layers;
		}
		break;
		case 2:
		{
			energy_kernel<EXCHNUM_BIG_GRID_SIZE, layers><<< gridSize, blockSize >>>(gridSizeZ, gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_energy.data,
																																							gpu_wang.data, gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																							normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_functions.data,
																																							Ndens, factor_output, update_rmm);
			if (update_rmm) {
				hipError_t error = hipGetLastError();
				if (error != hipSuccess) fprintf(stderr, "=!=!=!=!=====> CUDA ERROR <=====!=!=!=!=: %s\n", hipGetErrorString(error));
				
				calc_new_rmm<layers, EXCHNUM_BIG_GRID_SIZE><<<rmmGridSize, rmmBlockSize>>>(gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_wang.data,
																																										gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																										normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_rmm_output.data,
																																										factor_output, gpu_functions.data);
			}
			curr_cpu_layers = cpu_layers;
		}
		break;
	}
	

	/** CPU Accumulation */
	#ifndef _DEBUG
	if (!update_rmm)
	#endif
	{
		double energy_double = 0.0;		
		energy = gpu_energy;
		
		for (unsigned int i = 0; i < natoms; i++) {
			for (unsigned int j = 0; j < curr_cpu_layers[types.data[i]]; j++) {
				for (unsigned int k = 0; k < npoints; k++) {
					#ifdef ENERGY3D
					uint idx = index_from3d(threads, dim3(i, j, k));
					#else
					uint idx = index_from3d(dim3(threads.x, MAX_LAYERS, threads.y), dim3(i, j, k));
					#endif
					printf("idx: %i size: %i\n", idx, energy.elements());

					double energy_curr = energy.data[idx];
					printf("atomo: %i, capa: %i, punto: %i, valor: %.12e idx: %i\n", i, j, k, energy_curr, idx);
					energy_double += energy_curr;
					energy.data[0] += energy_curr;
				}
			}
		}
		printf("Energy (double): %.12e\n", energy_double);		
	}

	/** RMM update **/
	if (update_rmm) {
		HostMatrixFloat gpu_rmm_output_copy(gpu_rmm_output);
		
		uint rmm_idx = 0;
		for (uint func_i = 0; func_i < m; func_i++) {
			for (uint func_j = func_i; func_j < m; func_j++) {
				printf("rmm_output(%i): %.12e\n", rmm_idx, gpu_rmm_output_copy.data[rmm_idx]);
				cpu_rmm_output[rmm_idx] += gpu_rmm_output_copy.data[rmm_idx];
				rmm_idx++;
			}
		}
	}
	
	// calc_accum_cuda(gpu_energy, gpu_total_energy);

	// TODO: esta copia es redundante con la que hay en calc_acuum (esa es GPU<->GPU)
	// energy.copy_submatrix(gpu_total_energy, 1);
	// 
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) fprintf(stderr, "=!=!=!=!=====> CUDA ERROR <=====!=!=!=!=: %s\n", hipGetErrorString(error));
}


/************************************************** FUNCTIONS ****************************************/
#include "functions.h"

/************************************* DENSITY KERNEL ******************************/

#include "density.h"

/******************************** POT KERNEL ***********************************/

#include "pot.h"
