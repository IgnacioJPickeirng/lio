#include "hip/hip_runtime.h"
/* -*- mode: c -*- */
#include <cstdio>
#include "cuda_extra.h"
#include "../matrix.h"
#include "accum.h"
#include "exchnum.h"
#include "exchnum_constants.h"
#include "../timer.h"

#include <cassert>

using namespace G2G;
using namespace std;

/**
 * TODO: revisar distance / distance2 cuando sea necesario
 */

template <const uint* const curr_layers, uint grid_n>
__global__ void calc_new_rmm(const float3* atom_positions, const uint* types, const float3* point_positions,
														 const float* wang, const uint atoms_n, uint nco, uint3 num_funcs,
														 const uint* nuc, const uint* contractions, bool normalize, const float* factor_a, const float* factor_c,
														 const float* rmm, float* rmm_output, float* factors);


__device__ void calc_function(const uint3& num_funcs, const uint* nuc, const uint* contractions, const float3& point_position,
															const float3* atom_positions, const float* factor_a, const float* factor_c, uint big_func_index,
															bool normalize, float& func_value);

/***************************************** ENERGY KERNEL ******************************************/
#include "energy.h"

/*************************************** ACTUALIZACION DE RMM *************************************/
#include "rmm.h"


/**
 * Fortran interface
 */

/** TODO: ----- ARREGLAR ESTO ---------- */
#define FORTRAN_MAX_ATOMS 1845
#define FORTRAN_NG 900
#define FORTRAN_NL 10

/**
 * Parametros innecesarios: m (es sum(num_funcs))
 */
extern "C" void exchnum_gpu_(const unsigned int& norm, const unsigned int& natom, const double* r, const unsigned int* Iz, const unsigned int* Nuc,
														 const unsigned int& m, const unsigned int* ncont, const unsigned int* nshell, const double* c, const double* a,
														 double* RMM, const unsigned int& m18, const unsigned int& m5, const unsigned int& nco, double& Exc, const unsigned int& nopt,
														 const unsigned int& Iexch, const unsigned int& igrid,
														 const double* e, const double* e2, const double* e3,
														 const double* fort_wang, const double* fort_wang2, const double* fort_wang3,
														 const unsigned int& Ndens, const unsigned int& is_int3lu)
{
	Timer timer_exchnum;
	timer_exchnum.start();
	
	printf("<======= exchnum_gpu (from %s) ============>\n", is_int3lu ? "int3lu" : "SCF");
	printf("Ndens: %i\n", Ndens);
	uint3 num_funcs = make_uint3(nshell[0], nshell[1], nshell[2]);
	uint3 num_funcs_div = num_funcs / make_uint3(1, 3, 6);
	
	uint total_funcs = sum(num_funcs);
	uint total_funcs_div = sum(num_funcs_div);
	
	uint points = EXCHNUM_SMALL_GRID_SIZE;
	switch (igrid) {
		case 0: points = EXCHNUM_SMALL_GRID_SIZE; 	break;
		case 1: points = EXCHNUM_MEDIUM_GRID_SIZE;	break;
		case 2: points = EXCHNUM_BIG_GRID_SIZE;			break;
	}
	
	dim3 threads(natom, MAX_LAYERS, points);
	dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 gridSize3d = divUp(threads, blockSize);	
	
	HostMatrixFloat3 atom_positions(natom), point_positions(points);
	/* output_rmm size: TODO: divUp(m * (m - 1),2) */
	HostMatrixFloat factor_a(total_funcs, MAX_CONTRACTIONS), factor_c(total_funcs, MAX_CONTRACTIONS), wang(points);
	HostMatrixUInt types(natom), nuc(total_funcs_div), contractions(total_funcs_div);
	
	// REVISAR: nuc: imagen y dominio (especialmente por la parte de * 3 y * 6)

	printf("%i atoms\n", natom);
	for (unsigned int i = 0; i < natom; i++) {
		atom_positions.data[i] = make_float3(r[FORTRAN_MAX_ATOMS * 0 + i], r[i + FORTRAN_MAX_ATOMS * 1], r[i + FORTRAN_MAX_ATOMS * 2]);
		//printf("Pos(%i): %f %f %f, Types(%i): %i\n", i, atom_positions.data[i].x, atom_positions.data[i].y, atom_positions.data[i].z, i, Iz[i]);		
		types.data[i] = Iz[i] - 1;
	}
	
	printf("ns: %i, np: %i, nd: %i, Total_Funcs: %i\n", num_funcs.x, num_funcs.y, num_funcs.z, total_funcs);
	{
		uint inc = 1;
		uint i, j;
		for (i = 0, j = 0; i < total_funcs; i += inc, j++) {
			if (i == num_funcs.x) inc = 3;
			else if (i == num_funcs.x + num_funcs.y) inc = 6;

			//printf("i: %i, j: %i\n", i, j);
			//printf("Nuc(%i) = %i\n", i, Nuc[i] - 1);
			//printf("ncont(%i) = %i\n", i, ncont[i]);
			nuc.data[j] = Nuc[i] - 1;
			contractions.data[j] = ncont[i];
			
			for (unsigned int k = 0; k < ncont[i]; k++) {
				factor_a.data[j * MAX_CONTRACTIONS + k] = a[FORTRAN_NG * k + i];
				factor_c.data[j * MAX_CONTRACTIONS + k] = c[FORTRAN_NG * k + i];
				//printf("cont: %i, a: %f, c: %f\n", k, factor_a.data[j * MAX_CONTRACTIONS + k], factor_c.data[j * MAX_CONTRACTIONS + k]);
			}			
		}
	}
	
	HostMatrixFloat rmm;
	assert(Iexch == 1);
	printf("NCO: %i, M: %i, Iexch: %i\n", nco, total_funcs, Iexch);
	{
		if (Ndens == 1) {
			rmm.resize(m * m);
			uint k = 0;
			for (unsigned int i = 0; i < m; i++) {
				for (unsigned int j = i; j < m; j++) {
					rmm.data[k] = RMM[k];
					//printf("rmm(%i): %.30e\n", k, RMM[m5 + k - 1]);
					k++;
				}
			}
		}
		else {
			rmm.resize(m, nco);
			uint k = m18 - 1;
			for (unsigned int i = 0; i < m; i++) {
				for (unsigned int j = 0; j < nco; j++) {
					rmm.data[i * nco + j] = RMM[k];
					//printf("rmm(%i,%i): %.30e (%i)\n", i, j, RMM[k], k);
					k++;
				}
			}
		}
	}

	const double* real_e = NULL;
	const double* real_wang = NULL;
	switch (igrid) {
		case 0: real_e = e;  real_wang = fort_wang;  	break;
		case 1: real_e = e2; real_wang = fort_wang2; 	break;
		case 2: real_e = e3; real_wang = fort_wang3; 	break;		
	}

	printf("Puntos (grilla %i):\n", igrid);	
	for (unsigned int i = 0; i < points; i++) {
		wang.data[i] = real_wang[i];
		point_positions.data[i] = make_float3(real_e[0 * points + i], real_e[1 * points + i], real_e[2 * points + i]);
		//printf("wang: %f, e: (%f,%f,%f)\n", wang.data[i], point_positions.data[i].x, point_positions.data[i].y, point_positions.data[i].z);
	}
	
	HostMatrixDouble rmm_partial_out(m * m);
	rmm_partial_out.fill(0.0f);
	
	timer_exchnum.stop();
	printf("TIMER: exchnum_gpu:"); timer_exchnum.print(); printf("\n");	
		
	HostMatrixFloat energy(1);
	calc_energy(atom_positions, types, igrid, point_positions, energy, wang,
							Ndens, nco, num_funcs_div, nuc, contractions, norm, factor_a, factor_c, rmm, &RMM[m5-1],
							is_int3lu, threads, blockSize, gridSize3d);

	if (!is_int3lu) {
		/* update fortran variables */
		Exc = energy.data[0];
		printf("Exc: %f\n", energy.data[0]);
	}
}

/**
 * Host <-> CUDA Communication function
 */

void calc_energy(const HostMatrixFloat3& atom_positions, const HostMatrixUInt& types, uint grid_type,
								 const HostMatrixFloat3& point_positions, HostMatrixFloat& energy, const HostMatrixFloat& wang,
								 uint Ndens, uint nco, uint3 num_funcs, const HostMatrixUInt& nuc,
								 const HostMatrixUInt& contractions, bool normalize, const HostMatrixFloat& factor_a, const HostMatrixFloat& factor_c,
								 const HostMatrixFloat& rmm, double* cpu_rmm_output, bool update_rmm, const dim3& threads, const dim3& blockSize, const dim3& gridSize3d)
{	
	Timer timer_calc_energy;
	timer_calc_energy.start();
	
	const CudaMatrixFloat3 gpu_atom_positions(atom_positions);
	const CudaMatrixUInt gpu_types(types), gpu_nuc(nuc), gpu_contractions(contractions);
	
	uint gridSizeZ = gridSize3d.z;
	dim3 gridSize = dim3(gridSize3d.x, gridSize3d.y * gridSize3d.z, 1);
	
	uint m = num_funcs.x + num_funcs.y * 3 + num_funcs.z * 6;	
	uint small_m = sum(num_funcs);
	
	CudaMatrixFloat gpu_energy/*(threads.x * threads.y * threads.z)*/, gpu_total_energy, gpu_wang(wang), gpu_factor_a(factor_a), gpu_factor_c(factor_c),
									gpu_rmm(rmm), gpu_functions(m *  threads.x * threads.y * threads.z);
	CudaMatrixFloat3 gpu_point_positions(point_positions);
	
	printf("creando espacio para funcs output: size: %i (%i bytes) data: %i\n", gpu_functions.elements(), gpu_functions.bytes(), (bool)gpu_functions.data);	

	// optional update of RMM(M5)
	CudaMatrixFloat gpu_rmm_output;
	if (update_rmm) {
		// gpu_rmm_output.resize((m * (m + 1)) / 2);
		gpu_rmm_output.resize(m * m);
		printf("creando espacio para rmm output: size: %i (%i bytes) data: %i\n", gpu_rmm_output.elements(), gpu_rmm_output.bytes(), (bool)gpu_rmm_output.data);
	}
	#ifndef _DEBUG
	else
	#endif
		gpu_energy.resize(threads.x * threads.y * threads.z);
	
	printf("threads: %i %i %i, blockSize: %i %i %i, gridSize: %i %i %i\n", threads.x, threads.y, threads.z, blockSize.x, blockSize.y, blockSize.z, gridSize.x, gridSize.y / gridSizeZ, gridSizeZ);
	if (update_rmm) printf("GPU RMM SIZE: %i (%i bytes)\n", gpu_rmm_output.elements(), gpu_rmm_output.bytes());
	printf("energy data elements: %i data: %i\n", gpu_energy.elements(), (bool)gpu_energy.data);
	// TODO: update_rmm should be a template parameter
	const uint* curr_cpu_layers = NULL;
	
	float* factor_output = NULL;
	
	CudaMatrixFloat gpu_factor_output(threads.x * threads.y * threads.z);
	factor_output = gpu_factor_output.data;

	#if 0
	dim3 rmmThreads(divUp(m, 2), m + 1);
	dim3 rmmBlockSize(8,16);
	dim3 rmmGridSize = divUp(rmmThreads, rmmBlockSize);
	#else
	/*dim3 rmmThreads(m,m);
	dim3 rmmBlockSize(1);
	dim3 rmmGridSize(m,m);*/
	dim3 rmmThreads(m, m);
	dim3 rmmBlockSize(8,16);
	dim3 rmmGridSize = divUp(rmmThreads, rmmBlockSize);
	#endif
	
	printf("rmm threads: %i %i, blockSize: %i %i, gridSize: %i %i\n", rmmThreads.x, rmmThreads.y, rmmBlockSize.x, rmmBlockSize.y, rmmGridSize.x, rmmGridSize.y);		
	
	switch(grid_type) {
		case 0:
		{
			energy_kernel<EXCHNUM_SMALL_GRID_SIZE, layers2><<< gridSize, blockSize >>>(gridSizeZ, gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_energy.data,
																																								 gpu_wang.data, gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																								 normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_functions.data,
																																								 Ndens, factor_output, update_rmm);
			
			if (update_rmm) {				
				hipError_t error = hipGetLastError();
				if (error != hipSuccess) fprintf(stderr, "=!=!=!=!=====> CUDA ERROR <=====!=!=!=!=: %s\n", hipGetErrorString(error));
				
				//hipDeviceSynchronize();
				
				calc_new_rmm<layers2, EXCHNUM_SMALL_GRID_SIZE><<<rmmGridSize, rmmBlockSize>>>(gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_wang.data,
																																											gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																											normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_rmm_output.data,
																																											factor_output, gpu_functions.data);
				//hipDeviceSynchronize();
				
			}

			curr_cpu_layers = cpu_layers2;
		}
		break;
		case 1:
		{
			energy_kernel<EXCHNUM_MEDIUM_GRID_SIZE, layers><<< gridSize, blockSize >>>(gridSizeZ, gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_energy.data,
																																								 gpu_wang.data, gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																								 normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_functions.data, 
																																								 Ndens, factor_output, update_rmm);
			if (update_rmm) {
				hipError_t error = hipGetLastError();
				if (error != hipSuccess) fprintf(stderr, "=!=!=!=!=====> CUDA ERROR <=====!=!=!=!=: %s\n", hipGetErrorString(error));

				//hipDeviceSynchronize();
				calc_new_rmm<layers, EXCHNUM_MEDIUM_GRID_SIZE><<<rmmGridSize, rmmBlockSize>>>(gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_wang.data,
																																											 gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																											 normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_rmm_output.data,
																																											 factor_output, gpu_functions.data);
				//hipDeviceSynchronize();
				
			}

			curr_cpu_layers = cpu_layers;
		}
		break;
		case 2:
		{
			energy_kernel<EXCHNUM_BIG_GRID_SIZE, layers><<< gridSize, blockSize >>>(gridSizeZ, gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_energy.data,
																																							gpu_wang.data, gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																							normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_functions.data,
																																							Ndens, factor_output, update_rmm);
			if (update_rmm) {
				hipError_t error = hipGetLastError();
				if (error != hipSuccess) fprintf(stderr, "=!=!=!=!=====> CUDA ERROR <=====!=!=!=!=: %s\n", hipGetErrorString(error));
				
				//hipDeviceSynchronize();
				calc_new_rmm<layers, EXCHNUM_BIG_GRID_SIZE><<<rmmGridSize, rmmBlockSize>>>(gpu_atom_positions.data, gpu_types.data, gpu_point_positions.data, gpu_wang.data,
																																										gpu_atom_positions.width, nco, num_funcs, gpu_nuc.data, gpu_contractions.data,
																																										normalize, gpu_factor_a.data, gpu_factor_c.data, gpu_rmm.data, gpu_rmm_output.data,
																																										factor_output, gpu_functions.data);
				//hipDeviceSynchronize();
				
			}
			curr_cpu_layers = cpu_layers;
		}
		break;
	}
	

	/** CPU Accumulation */
	#ifndef _DEBUG
	if (!update_rmm)
	#endif
		energy = gpu_energy;
	
	HostMatrixFloat gpu_rmm_output_copy(gpu_rmm_output);
	
	//hipDeviceSynchronize();

	double energy_double = 0.0;

	#ifndef _DEBUG
	if (!update_rmm)
	#endif
	{
		for (unsigned int i = 0; i < threads.x; i++) {
			for (unsigned int j = 0; j < curr_cpu_layers[types.data[i]]; j++) {
				for (unsigned int k = 0; k < threads.z; k++) {
					uint idx = index_from3d(threads, dim3(i, j, k));
					printf("idx: %i size: %i\n", idx, energy.elements());

					double energy_curr = energy.data[idx];
					printf("atomo: %i, capa: %i, punto: %i, valor: %.12e idx: %i\n", i, j, k, energy_curr, idx);
					energy_double += energy_curr;
					energy.data[0] += energy_curr;
				}
			}
		}
		printf("Energy (double): %.12e\n", energy_double);		
	}
	
	if (update_rmm) {
		uint rmm_idx = 0;
		for (uint func_i = 0; func_i < m; func_i++) {
			for (uint func_j = func_i; func_j < m; func_j++) {
				printf("rmm_output(%i): %.12e\n", rmm_idx, gpu_rmm_output_copy.data[rmm_idx]);
				cpu_rmm_output[rmm_idx] += gpu_rmm_output_copy.data[rmm_idx];
				rmm_idx++;
			}
		}
	}
	
	// calc_accum_cuda(gpu_energy, gpu_total_energy);

	// TODO: esta copia es redundante con la que hay en calc_acuum (esa es GPU<->GPU)
	// energy.copy_submatrix(gpu_total_energy, 1);
	// 
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) fprintf(stderr, "=!=!=!=!=====> CUDA ERROR <=====!=!=!=!=: %s\n", hipGetErrorString(error));
		
//	hipDeviceSynchronize();		
	timer_calc_energy.stop();
//	printf("TIMER: calc_energy:"); timer_calc_energy.print(); printf("\n");
}


/************************************************** FUNCTIONS ****************************************/
#include "functions.h"

/************************************* DENSITY KERNEL ******************************/

#include "density.h"

/******************************** POT KERNEL ***********************************/

#include "pot.h"
