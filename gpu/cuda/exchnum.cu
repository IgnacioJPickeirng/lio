#include "hip/hip_runtime.h"
/* -*- mode: c -*- */
#include <cassert>
#include <iostream>
#include <fstream>
#include <map>
#include <string>
#include "../common.h"
#include "../init.h"
#include "cuda_extra.h"
#include "../matrix.h"
#include "gpu_variables.h"
#include "../timer.h"
#include "double.h"
#include "../partition.h"

#define OLD_DENSITY_KERNEL 1

/** KERNELS **/
#include "functions.h"
#include "energy.h"
#include "rmm.h"
#include "force.h"
#include "weight.h"

/** CPU Kernels **/
#include "../exchnum.cpp"

using namespace G2G;
using namespace std;


#define COMPUTE_RMM 					0
#define COMPUTE_ENERGY_ONLY		1
#define COMPUTE_ENERGY_FORCE	2
#define COMPUTE_FORCE_ONLY		3


/*******************************
 * Cube Functions
 *******************************/
void gpu_compute_group_functions(void)
{
	cout << "<===== computing functions ========>" << endl;
	CudaMatrixFloat3 points_position_gpu;
	CudaMatrixFloat2 factor_ac_gpu;
	CudaMatrixUInt nuc_gpu;
	CudaMatrixUInt contractions_gpu;
	
	Timer t1;
	t1.sync();
	t1.start();
	
	for (list<PointGroup>::iterator it = final_partition.begin(); it != final_partition.end(); ++it) {
		PointGroup& group = *it;
		/** Load points from group **/
		{
			HostMatrixFloat3 points_position_cpu(group.number_of_points, 1);
						
			uint i = 0;		
			for (list<Point>::const_iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
				points_position_cpu.get(i) = make_float3(p->position.x, p->position.y, p->position.z);
			}
			points_position_gpu = points_position_cpu;
		}
		
		/* Load group functions */
		uint group_m = group.s_functions + group.p_functions * 3 + group.d_functions * 6;
		uint group_spd = group.s_functions + group.p_functions + group.d_functions;
		uint4 group_functions = make_uint4(group.s_functions, group.p_functions, group.d_functions, group_m);
		{
			HostMatrixFloat2 factor_ac_cpu(group_spd, MAX_CONTRACTIONS);
			HostMatrixUInt nuc_cpu(group_spd, 1), contractions_cpu(group_spd, 1);
			
			uint i = 0;
			for (set<uint>::const_iterator func = group.functions.begin(); func != group.functions.end(); ++func, ++i) {
				nuc_cpu.get(i) = fortran_vars.nucleii.get(*func) - 1;
				contractions_cpu.get(i) = fortran_vars.contractions.get(*func);
				assert(contractions_cpu.get(i) <= MAX_CONTRACTIONS);
				
				for (unsigned int k = 0; k < contractions_cpu.get(i); k++)
					factor_ac_cpu.get(i, k) = make_float2(fortran_vars.a_values.get(*func, k), fortran_vars.c_values.get(*func, k));
			}

			factor_ac_gpu = factor_ac_cpu;
			nuc_gpu = nuc_cpu;
			contractions_gpu = contractions_cpu;
		}
		
		/** Compute Functions **/		
    group.function_values.resize(COALESCED_DIMENSION(group.number_of_points), group_functions.w);
    if (fortran_vars.do_forces) group.gradient_values.resize(COALESCED_DIMENSION(group.number_of_points), group_functions.w);
		
		dim3 threads(group.number_of_points);
		dim3 threadBlock(FUNCTIONS_BLOCK_SIZE);
		dim3 threadGrid = divUp(threads, threadBlock);		

		//cout << "points: " << threads.x << " " << threadGrid.x << " " << threadBlock.x << endl;
		
		if (fortran_vars.do_forces)
			gpu_compute_functions<true><<<threadGrid, threadBlock>>>(points_position_gpu.data, group.number_of_points, contractions_gpu.data, factor_ac_gpu.data, nuc_gpu.data, group.function_values.data, group.gradient_values.data, group_functions, group_spd);
		else
			gpu_compute_functions<false><<<threadGrid, threadBlock>>>(points_position_gpu.data, group.number_of_points, contractions_gpu.data, factor_ac_gpu.data, nuc_gpu.data, group.function_values.data, group.gradient_values.data, group_functions, group_spd);

		cudaAssertNoError("compute_functions");

#if 0
		if (fortran_vars.grid_type == BIG_GRID) {
			cout << "s_funcs: " << group.s_functions << " p_funcs " << group.p_functions << " d_funcs " << group.d_functions << endl;
			HostMatrixFloat functions_cpu(group.function_values);
			HostMatrixFloat3 gradients_cpu(group.gradient_values);
			uint i = 0;		
			for (list<Point>::const_iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
				uint func_idx = 0;
				for (set<uint>::const_iterator func = group.functions.begin(); func != group.functions.end(); ++func, ++func_idx) {
					if (fortran_vars.nucleii.get(*func) - 1 != 0) continue;
					if (func_idx < group.s_functions)
						cout << "* point (" << p->atom << "," << p->shell << "," << p->point << ") - Fg(" << *func << ")=" << gradients_cpu.get(func_idx, i).x << " "  << gradients_cpu.get(func_idx, i).y << " " << gradients_cpu.get(func_idx, i).z << " F " << functions_cpu.get(func_idx, i) << " " << func_idx << endl;
					else if (func_idx < group.p_functions + group.s_functions) {
						uint p_idx = 3 * (func_idx - group.s_functions) + group.s_functions;
						for (uint j = 0; j < 3; j++)
							cout << "* point (" << p->atom << "," << p->shell << "," << p->point << ") - Fg(" << *func << ")=" << gradients_cpu.get(p_idx + j, i).x << " "  << gradients_cpu.get(p_idx + j, i).y << " " << gradients_cpu.get(p_idx + j, i).z << " F " << functions_cpu.get(p_idx + j, i) << " " << p_idx + j << endl;
					}
					else {
						uint s_idx = group.s_functions + group.p_functions * 3 + 6 * (func_idx - group.s_functions - group.p_functions);
						for (uint j = 0; j < 6; j++)
							cout << "* point (" << p->atom << "," << p->shell << "," << p->point << ") - Fg(" << *func << ")=" << gradients_cpu.get(s_idx + j, i).x << " "  << gradients_cpu.get(s_idx + j, i).y << " " << gradients_cpu.get(s_idx + j, i).z << " F " << functions_cpu.get(s_idx + j, i) << " " << s_idx + j << endl;

					}
//				cout << "* point " << p->position.x << " " << p->position.y << " " << p->position.z << " " << functions_cpu.get(p_idx, i) << endl;
				}
			}
		}
#endif
	}	
	
	t1.sync();
	t1.stop();
	cout << "TIMER: funcs: " << t1 << endl;
}

/*******************************
 * Cube Weights
 *******************************/

void gpu_compute_group_weights(PointGroup& group)
{
  CudaMatrixFloat4 point_positions_gpu;
  {
    HostMatrixFloat4 points_positions_cpu(group.number_of_points, 1);

		uint i = 0;
		for (list<Point>::const_iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
			points_positions_cpu.get(i) = make_float4(p->position.x, p->position.y, p->position.z, p->atom);
		}
		point_positions_gpu = points_positions_cpu;
	}

  CudaMatrixFloat weights_gpu(group.number_of_points);
  dim3 threads(group.number_of_points);
  dim3 blockSize(WEIGHT_BLOCK_SIZE);
  dim3 gridSize = divUp(threads, blockSize);
  gpu_compute_weights<<<gridSize,blockSize>>>(group.number_of_points, point_positions_gpu.data, weights_gpu.data);
  cudaAssertNoError("compute_weights");

  HostMatrixFloat weights_cpu(weights_gpu);
  uint i = 0;
  for (list<Point>::iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
    p->weight *= weights_cpu.get(i);
  }
}

/********************************
 * Solve Cubes
 ********************************/
extern "C" void gpu_solve_groups_(uint& computation_type, double* fort_energy_ptr, double* fort_forces_ptr)
{
	cout << "<================ calculo de: [";
	switch(computation_type) {
		case COMPUTE_ENERGY_ONLY: cout << "energia"; break;
		case COMPUTE_RMM: cout << "rmm"; break;
		case COMPUTE_FORCE_ONLY: cout << "fuerzas"; break;
		case COMPUTE_ENERGY_FORCE: cout << "energia+fuerzas"; break;
	}
	cout << "] ==========>" << endl;
	
	Timer t_total;
	t_total.sync();
	t_total.start();
		
	/*** Computo sobre cada cubo ****/
	CudaMatrixFloat point_weights_gpu;
	CudaMatrixFloat rdm_gpu, rdmt_gpu;
	CudaMatrixUInt nuc_gpu;

  Timer t_density, t_rmm, t_forces;
  Timer t_cpu;

	FortranMatrix<double> fort_forces(fort_forces_ptr, fortran_vars.atoms, 3, FORTRAN_MAX_ATOMS);
	
	double total_energy = 0.0;
		
	for (list<PointGroup>::const_iterator it = final_partition.begin(); it != final_partition.end(); ++it) {
		const PointGroup& group = *it;
    //cout << "group is " << (group.is_sphere ? "sphere" : "cube") << endl;
				
		/** Load points from group **/
    #if !CPU_KERNELS
		{
    #endif
			HostMatrixFloat point_weights_cpu(group.number_of_points, 1);

			uint i = 0;		
			for (list<Point>::const_iterator p = group.points.begin(); p != group.points.end(); ++p, ++i) {
				point_weights_cpu.get(i) = p->weight;
			}
			point_weights_gpu = point_weights_cpu;
    #if !CPU_KERNELS
		}
    #endif
		
		/** Load functions from group **/
		uint group_m = group.s_functions + group.p_functions * 3 + group.d_functions * 6;
		uint4 group_functions = make_uint4(group.s_functions, group.p_functions, group.d_functions, group_m);
		
		/* load RDM */
    #if !CPU_KERNELS
		{
    #endif
			HostMatrixFloat rdm_cpu(COALESCED_DIMENSION(group_m), fortran_vars.nco);
      HostMatrixFloat rdmt_cpu;

      if (computation_type == COMPUTE_ENERGY_FORCE || computation_type == COMPUTE_FORCE_ONLY)
        rdmt_cpu.resize(COALESCED_DIMENSION(fortran_vars.nco), group_m);

			for (unsigned int i = 0; i < fortran_vars.nco; i++) {
				uint j = 0;
				for (set<uint>::const_iterator func = group.functions.begin(); func != group.functions.end(); ++func) {
					if (*func < fortran_vars.s_funcs) {
						rdm_cpu.get(j, i) = fortran_vars.rmm_input.get(*func, i);
            if (rdmt_cpu.is_allocated()) rdmt_cpu.get(i, j) = rdm_cpu.get(j, i);
						j++;
					}
					else if (*func < (fortran_vars.s_funcs + fortran_vars.p_funcs * 3)) {
						for (uint k = 0; k < 3; k++, j++) {
              rdm_cpu.get(j, i) = fortran_vars.rmm_input.get(*func + k, i);
              if (rdmt_cpu.is_allocated()) rdmt_cpu.get(i, j) = rdm_cpu.get(j, i);
            }
					}
					else {
						for (uint k = 0; k < 6; k++, j++) {
              rdm_cpu.get(j, i) = fortran_vars.rmm_input.get(*func + k, i);
              if (rdmt_cpu.is_allocated()) rdmt_cpu.get(i, j) = rdm_cpu.get(j, i);
            }
					}
				}
			}
			rdm_gpu = rdm_cpu;
      if (rdmt_cpu.is_allocated()) rdmt_gpu = rdmt_cpu;
    #if !CPU_KERNELS
		}
    #endif
							
		dim3 threads(group.number_of_points);
		dim3 threadBlock, threadGrid;
		threadBlock = dim3(DENSITY_BLOCK_SIZE);
		threadGrid = divUp(threads, threadBlock);
    //cout << "density/energy threads: " << threads.x << " blocks: " << threadGrid.x << " blockSize: " << threadBlock.x << endl;

		/* compute energy */
		if (computation_type == COMPUTE_ENERGY_ONLY) {
      #if CPU_KERNELS
      HostMatrixFloat energy_cpu(group.number_of_points);
      HostMatrixFloat function_values_cpu(group.function_values.height, group.function_values.width);
      function_values_cpu.copy_transpose(group.function_values);

      t_cpu.start_and_sync();
      cpu_compute_density_forces<true, false>(energy_cpu.data, point_weights_cpu.data, group.number_of_points, rdm_cpu.data,
        NULL, function_values_cpu.data, NULL, NULL, NULL, 0, group_m, t_density, t_rmm);
      t_cpu.pause_and_sync();
      #else
      t_density.start_and_sync();
			CudaMatrixFloat energy_gpu(group.number_of_points);
			gpu_compute_density<true, false><<<threadGrid, threadBlock>>>(energy_gpu.data, NULL, point_weights_gpu.data, group.number_of_points,
                                                                    rdm_gpu.data, group.function_values.data, group_m, NULL);
			cudaAssertNoError("compute_density");
      t_density.pause_and_sync();
      HostMatrixFloat energy_cpu(energy_gpu);
      #endif
			
			for (uint i = 0; i < group.number_of_points; i++) { total_energy += energy_cpu.get(i); }
		}
		/* compute necessary factor **/
		else if (computation_type == COMPUTE_RMM) {
      #if CPU_KERNELS

      HostMatrixFloat function_values_cpu(group.function_values.height, group.function_values.width);
      function_values_cpu.copy_transpose(group.function_values);
      HostMatrixFloat rmm_output_cpu(COALESCED_DIMENSION(group_m), group_m);
      t_cpu.start_and_sync();
      cpu_compute_density_forces<false, false>(NULL, point_weights_cpu.data, group.number_of_points, rdm_cpu.data,
        rmm_output_cpu.data, function_values_cpu.data, NULL, NULL, NULL, 0, group_m, t_density, t_rmm);
      t_cpu.pause_and_sync();
      
      #else

			CudaMatrixFloat rmm_factor_gpu(group.number_of_points);
      t_density.start_and_sync();
			gpu_compute_density<false, false><<<threadGrid, threadBlock>>>(NULL, rmm_factor_gpu.data, point_weights_gpu.data, group.number_of_points,
                                                                     rdm_gpu.data, group.function_values.data, group_m, NULL);
			cudaAssertNoError("compute_density");
      t_density.pause_and_sync();

			/*** Compute RMM update ***/
			threads = dim3(group_m, group_m);
			threadBlock = dim3(RMM_BLOCK_SIZE_XY, RMM_BLOCK_SIZE_XY);
			threadGrid = divUp(threads, threadBlock);

      CudaMatrixFloat rmm_output_gpu(COALESCED_DIMENSION(group_m), group_m);
      t_rmm.start_and_sync();
			gpu_update_rmm<<<threadGrid, threadBlock>>>(rmm_factor_gpu.data, group.number_of_points, rmm_output_gpu.data, group.function_values.data, group_m);
			cudaAssertNoError("update_rmm");
      t_rmm.pause_and_sync();

			HostMatrixFloat rmm_output_cpu(rmm_output_gpu);
      #endif

      /*** Contribute this RMM to the total RMM ***/
      uint small_fi = 0;

			for (set<uint>::iterator it_fi = group.functions.begin(); it_fi != group.functions.end(); ++it_fi) {
				uint fi_advance;
				if (*it_fi < fortran_vars.s_funcs) fi_advance = 1;
				else if (*it_fi < fortran_vars.s_funcs + fortran_vars.p_funcs * 3) fi_advance = 3;
				else fi_advance = 6;
				
				for (uint i = 0; i < fi_advance; i++) {

          uint small_fj = 0;
					for (set<uint>::iterator it_fj = group.functions.begin(); it_fj != group.functions.end(); ++it_fj) {
						uint fj_advance;
						if (*it_fj < fortran_vars.s_funcs) fj_advance = 1;
						else if (*it_fj < fortran_vars.s_funcs + fortran_vars.p_funcs * 3) fj_advance = 3;
						else fj_advance = 6;
					
						for (uint j = 0; j < fj_advance; j++) {
							uint fi = *it_fi + i; uint fj = *it_fj + j;
							if (fi > fj) continue;
							uint big_index = (fi * fortran_vars.m - (fi * (fi - 1)) / 2) + (fj - fi);
              fortran_vars.rmm_output.get(big_index) += rmm_output_cpu.get(small_fi, small_fj + small_fi);
              small_fj++;
						}					
					}
          small_fi++;
				}
			}
		}
		/* compute forces */
		else {
      map<uint, uint> nuc_mapping;
      #if !CPU_KERNELS
			{
      #endif
				HostMatrixUInt nuc_cpu(group_m, 1);
				uint i = 0;
        uint small_atom_idx = 0;

        for (set<uint>::iterator func = group.functions.begin(); func != group.functions.end(); ++func) {
          uint f_advance;
          if (*func < fortran_vars.s_funcs) f_advance = 1;
          else if (*func < fortran_vars.s_funcs + fortran_vars.p_funcs * 3) f_advance = 3;
          else f_advance = 6;

          for (uint j = 0; j < f_advance; j++, i++) {
            uint big_atom_idx = fortran_vars.nucleii.get(*func) - 1;
            if (nuc_mapping.find(big_atom_idx) == nuc_mapping.end()) {
              nuc_mapping[big_atom_idx] = small_atom_idx;
              small_atom_idx++;
            }
            nuc_cpu.get(i) = nuc_mapping[big_atom_idx];
          }
        }
				nuc_gpu = nuc_cpu;
      #if !CPU_KERNELS
			}
      #endif

      #if CPU_KERNELS
      HostMatrixFloat energy_cpu;;
      HostMatrixFloat4 forces_cpu(group.nucleii.size());

      if (computation_type == COMPUTE_ENERGY_FORCE) {
        energy_cpu.resize(group.number_of_points);
        HostMatrixFloat function_values_cpu(group.function_values.height, group.function_values.width);
        function_values_cpu.copy_transpose(group.function_values);
        HostMatrixFloat4 gradient_values_cpu(group.gradient_values);

        t_cpu.start_and_sync();
        cpu_compute_density_forces<true, true>(energy_cpu.data, point_weights_cpu.data, group.number_of_points, rdm_cpu.data, NULL,
          function_values_cpu.data, gradient_values_cpu.data, forces_cpu.data, nuc_cpu.data, group.nucleii.size(), group_m, t_density, t_rmm);
        t_cpu.pause_and_sync();

        for (uint i = 0; i < group.number_of_points; i++) { total_energy += energy_cpu.get(i); }
      }
      else {
        HostMatrixFloat function_values_cpu(group.function_values.height, group.function_values.width);
        function_values_cpu.copy_transpose(group.function_values);
        HostMatrixFloat4 gradient_values_cpu(group.gradient_values);
        
        t_cpu.start_and_sync();
        cpu_compute_density_forces<false,true>(NULL, point_weights_cpu.data, group.number_of_points, rdm_cpu.data, NULL,
          function_values_cpu.data, gradient_values_cpu.data, forces_cpu.data, nuc_cpu.data, group.nucleii.size(), group_m, t_density, t_rmm);
        t_cpu.pause_and_sync();
      }
      #else

			CudaMatrixFloat force_factor_gpu(group.number_of_points);
			CudaMatrixFloat energy_gpu;
      CudaMatrixFloat w_gpu(COALESCED_DIMENSION(group.number_of_points), fortran_vars.nco);

			/* energy may be needed at this step */
			CudaMatrixFloat4 density_deriv(COALESCED_DIMENSION(group.number_of_points), group.nucleii.size());
			if (computation_type == COMPUTE_ENERGY_FORCE) {
				energy_gpu.resize(group.number_of_points);
        t_density.start_and_sync();

				gpu_compute_density<true, true><<<threadGrid, threadBlock>>>(energy_gpu.data, force_factor_gpu.data, point_weights_gpu.data, group.number_of_points,
                                                               rdm_gpu.data, group.function_values.data, group_m, w_gpu.data);
        cudaAssertNoError("compute_density");
        t_density.pause_and_sync();

				HostMatrixFloat energy_cpu(energy_gpu);
				for (uint i = 0; i < group.number_of_points; i++) { total_energy += energy_cpu.get(i); }
			}
			else {
        t_density.start_and_sync();
				gpu_compute_density<false, true><<<threadGrid, threadBlock>>>(energy_gpu.data, force_factor_gpu.data, point_weights_gpu.data, group.number_of_points,
                                                                rdm_gpu.data, group.function_values.data, group_m, w_gpu.data);
        t_density.pause_and_sync();
        cudaAssertNoError("compute_density");
      }

      threadBlock = dim3(DENSITY_DERIV_BLOCK_SIZE);
      threadGrid = divUp(threads, threadBlock);

      t_density.start_and_sync();
			gpu_compute_density_derivs<<<threadGrid, threadBlock>>>(group.number_of_points, rdmt_gpu.data, group.gradient_values.data, density_deriv.data, nuc_gpu.data,
                                                              group.nucleii.size(), group_m, w_gpu.data);
      t_density.pause_and_sync();
      cudaAssertNoError("compute_density_deriv");

			threads = dim3(group.nucleii.size());
			threadBlock = dim3(FORCE_BLOCK_SIZE);
			threadGrid = divUp(threads, threadBlock);

			CudaMatrixFloat4 forces_gpu(group.nucleii.size());
      t_forces.start_and_sync();
			gpu_compute_forces<<<threadGrid, threadBlock>>>(group.number_of_points, force_factor_gpu.data, density_deriv.data, forces_gpu.data, group.nucleii.size());
      t_forces.pause_and_sync();
			cudaAssertNoError("gpu_compute_forces");
      HostMatrixFloat4 forces_cpu(forces_gpu);
      #endif

			for (map<uint, uint>::iterator nuc_it = nuc_mapping.begin(); nuc_it != nuc_mapping.end(); ++nuc_it) {
				float4 atom_force = forces_cpu.get(nuc_it->second);
        //cout << "atom force: " << atom_force.x << " " << atom_force.y << " " << atom_force.z << endl;
				fort_forces.get(nuc_it->first, 0) += atom_force.x;
				fort_forces.get(nuc_it->first, 1) += atom_force.y;
				fort_forces.get(nuc_it->first, 2) += atom_force.z;
      }
		}
	}
		
	/** pass results to fortran */
	if (computation_type == COMPUTE_ENERGY_ONLY || computation_type == COMPUTE_ENERGY_FORCE) {
		cout << "total energy: " << total_energy << endl;
		*fort_energy_ptr = total_energy;
	}
	t_total.stop_and_sync();

	cout << "TIMER: gpu_solve_cubes " << t_total << endl;
  cout << "TIMER: density/energy " << t_density << endl;
  cout << "TIMER: forces " << t_forces << endl;
  cout << "TIMER: rmm: " << t_rmm << endl;
  #if CPU_KERNELS
  cout << "TIMER: cpu: " << t_cpu << endl;
  #endif
}
