#include "hip/hip_runtime.h"
/* -*- mode: c -*- */
#include <iostream>
#include <fstream>
#include <map>
#include <string>
#include <hip/hip_math_constants.h>
#include "../common.h"
#include "../init.h"
#include "cuda_extra.h"
#include "../matrix.h"
#include "../timer.h"
#include "../partition.h"
#include "../scalar_vector_types.h"
#include "../global_memory_pool.h"

namespace G2G {
#if FULL_DOUBLE
texture<int2, 2, hipReadModeElementType> rmm_input_gpu_tex;
texture<int2, 2, hipReadModeElementType> rmm_input_gpu_tex2;
//texture<int2, hipTextureType2D, hipReadModeElementType> qmmm_F_values_tex;
#else
texture<float, 2, hipReadModeElementType> rmm_input_gpu_tex;
texture<float, 2, hipReadModeElementType> rmm_input_gpu_tex2;
//texture<float, hipTextureType2D, hipReadModeElementType> qmmm_F_values_tex;
#endif
/** KERNELS **/
#include "gpu_variables.h"
#include "kernels/pot.h"
#include "kernels/potop.h"
#include "kernels/accumulate_point.h"
#include "kernels/energy.h"
#include "kernels/energy_open.h"
#include "kernels/energy_derivs.h"
#include "kernels/rmm.h"
#include "kernels/weight.h"
#include "kernels/functions.h"
#include "kernels/force.h"
#include "kernels/transpose.h"
#include "kernels/qmmm.h"

using std::cout;
using std::endl;
using std::list;

// Host function to set the constant
void gpu_set_variables(void) {
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_normalization_factor), &fortran_vars.normalization_factor, sizeof(fortran_vars.normalization_factor), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_atoms), &fortran_vars.atoms, sizeof(fortran_vars.atoms), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_Iexch), &fortran_vars.iexch, sizeof(fortran_vars.iexch), 0, hipMemcpyHostToDevice);
  cudaAssertNoError("set_gpu_variables");
}

template<class scalar_type>
void gpu_set_gamma_arrays() {

  // Cast STR/FAC to appropriate type (float/double)
  HostMatrix<scalar_type> h_str(880,22), h_fac(17);
  for (uint i = 0; i < 880; i++) {
    for (uint j = 0; j < 22; j++) {
      h_str(i,j) = fortran_vars.str(i,j);
    }
  }
  for (uint i = 0; i < 17; i++) {
    h_fac(i) = fortran_vars.fac(i);
  }

  scalar_type* d_str_ptr;
  hipMalloc((void**)&d_str_ptr,880*22*sizeof(scalar_type));
  // STR data host->device
  hipMemcpy(d_str_ptr,h_str.data,h_str.bytes(),hipMemcpyHostToDevice);
  // STR device pointer h->d
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_str),&d_str_ptr,sizeof(gpu_str),0,hipMemcpyHostToDevice);

  // FAC data h->d
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_fac),h_fac.data,h_fac.bytes(),0,hipMemcpyHostToDevice);

  /*scalar_type* d_gamma_ptr;
  hipMalloc((void**)&d_gamma_ptr,GAMMA_LENGTH*6*sizeof(scalar_type));

  dim3 pg_threads(GAMMA_LENGTH,6);
  dim3 pg_blockSize(32,6);
  dim3 pg_gridSize(divUp(pg_threads,pg_blockSize));
  precompute_gamma<scalar_type><<<pg_gridSize,pg_blockSize>>>(GAMMA_LENGTH,GAMMA_INC,d_gamma_ptr);

  qmmm_F_values_tex.normalized = false;
  qmmm_F_values_tex.filterMode = hipFilterModeLinear;
  hipMallocArray(&gammaArray,&qmmm_F_values_tex.channelDesc,GAMMA_LENGTH,6);
  hipMemcpyToArray(gammaArray,0,0,d_gamma_ptr,sizeof(scalar_type)*GAMMA_LENGTH*6,hipMemcpyDeviceToDevice);
  hipFree(d_gamma_ptr);
  // Don't need STR past this point if precomputing F(m,U)
  // Can free STR on-device here*/

  cudaAssertNoError("gpu_set_gamma_arrays");
}

template<class T> void gpu_set_atom_positions(const HostMatrix<T>& m) {
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_atom_positions), m.data, m.bytes(), 0, hipMemcpyHostToDevice);
}
template<class T, class U> void gpu_set_clatoms(const HostMatrix<T>& m_pos, const HostMatrix<U>& m_charge) {
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_clatoms), &fortran_vars.clatoms, sizeof(fortran_vars.clatoms), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_clatom_positions), m_pos.data, m_pos.bytes(), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_clatom_charges), m_charge.data, m_charge.bytes(), 0, hipMemcpyHostToDevice);
}

#if FULL_DOUBLE
template void gpu_set_gamma_arrays<double>( void );
#else
template void gpu_set_gamma_arrays<float>( void );
#endif
template void gpu_set_atom_positions<float3>(const HostMatrix<float3>& m);
template void gpu_set_atom_positions<double3>(const HostMatrix<double3>& m);
template void gpu_set_clatoms<float3,float>(const HostMatrix<float3>& m_pos, const HostMatrix<float>& m_charge);
template void gpu_set_clatoms<double3,double>(const HostMatrix<double3>& m_pos, const HostMatrix<double>& m_charge);
//template<class scalar_type,true> __global__ void gpu_update_rmm(scalar_type* factors, uint points, scalar_type* rmm, scalar_type* function_values, uint m);
//template<class scalar_type,false> __global__ void gpu_update_rmm(scalar_type* factors, uint points, scalar_type* rmm, scalar_type* function_values, uint m);

template<class scalar_type>
void PointGroup<scalar_type>::solve(Timers& timers, bool compute_rmm, bool lda, bool compute_forces,
    bool compute_energy, double& energy,double& energy_i, double& energy_c, double& energy_c1,
    double& energy_c2, double* fort_forces_ptr, bool open){
  if(open) {
    solve_opened(timers, compute_rmm, lda, compute_forces, compute_energy, energy, energy_i, energy_c, energy_c1,
        energy_c2, fort_forces_ptr);
  }
  else {
    solve_closed(timers, compute_rmm, lda, compute_forces, compute_energy, energy, fort_forces_ptr);
  }

}

template<class scalar_type>
void PointGroup<scalar_type>::solve_closed(Timers& timers, bool compute_rmm, bool lda, bool compute_forces, bool compute_energy, double& energy, double* fort_forces_ptr){
  //uint max_used_memory = 0;

  /*** Computo sobre cada cubo ****/
  CudaMatrix<scalar_type> point_weights_gpu;
  FortranMatrix<double> fort_forces(fort_forces_ptr, fortran_vars.atoms, 3, fortran_vars.max_atoms);

  /** Compute this group's functions **/
  timers.functions.start_and_sync();
  compute_functions(compute_forces, !lda); //<<<===============
  timers.functions.pause_and_sync();

  uint group_m = total_functions();

  timers.density.start_and_sync();
  /** Load points from group **/
  HostMatrix<scalar_type> point_weights_cpu(number_of_points, 1);

  uint i = 0;
  for (list<Point>::const_iterator p = points.begin(); p != points.end(); ++p, ++i) {
    point_weights_cpu(i) = p->weight;
  }
  point_weights_gpu = point_weights_cpu;

//<<===========================>>//
  dim3 threadBlock, threadGrid;
  /* compute density/factors */

  const int block_height= divUp(group_m,2*DENSITY_BLOCK_SIZE);

  threadBlock = dim3(DENSITY_BLOCK_SIZE,1,1); // Hay que asegurarse que la cantidad de funciones este en rango
  threadGrid = dim3(number_of_points,block_height,1);

  CudaMatrix<scalar_type> factors_gpu;

  CudaMatrix<scalar_type> partial_densities_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dxyz_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd1_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd2_gpu;

  /*
   **********************************************************************
   * Transposiciones de matrices para la coalescencia mejorada en density
   **********************************************************************
   */


  CudaMatrix<scalar_type>   function_values_transposed_gpu;
  CudaMatrix<vec_type<scalar_type,4> > gradient_values_transposed_gpu;
  CudaMatrix<vec_type<scalar_type,4> > hessian_values_transposed_gpu;

  int transposed_width = COALESCED_DIMENSION(number_of_points);

  function_values_transposed_gpu.resize(group_m, COALESCED_DIMENSION(number_of_points));
  if (fortran_vars.do_forces || fortran_vars.gga)
      gradient_values_transposed_gpu.resize( group_m,COALESCED_DIMENSION(number_of_points));
  if (fortran_vars.gga)
      hessian_values_transposed_gpu.resize((group_m) * 2, COALESCED_DIMENSION(number_of_points));

  #define BLOCK_DIM 16
  dim3 transpose_grid(transposed_width / BLOCK_DIM, divUp((group_m),BLOCK_DIM));
  dim3 transpose_threads(BLOCK_DIM, BLOCK_DIM, 1);

  transpose<<<transpose_grid, transpose_threads>>> (function_values_transposed_gpu.data, function_values.data,  COALESCED_DIMENSION(number_of_points),group_m   );

  if (fortran_vars.do_forces || fortran_vars.gga)
    transpose_vec<<<transpose_grid, transpose_threads>>> (gradient_values_transposed_gpu.data, gradient_values.data, COALESCED_DIMENSION(number_of_points), group_m );

  transpose_grid=dim3(transposed_width / BLOCK_DIM, divUp((group_m)*2, BLOCK_DIM), 1);

  if (fortran_vars.gga)
    transpose_vec<<<transpose_grid, transpose_threads>>> (hessian_values_transposed_gpu.data, hessian_values.data, COALESCED_DIMENSION(number_of_points), (group_m)*2);


  partial_densities_gpu.resize(COALESCED_DIMENSION(number_of_points), block_height);
  dxyz_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height);
  dd1_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height );
  dd2_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height );

  const dim3 threadGrid_accumulate(divUp(number_of_points,DENSITY_ACCUM_BLOCK_SIZE),1,1);
  const dim3 threadBlock_accumulate(DENSITY_ACCUM_BLOCK_SIZE,1,1);

  if (compute_rmm || compute_forces) factors_gpu.resize(number_of_points);

  HostMatrix<scalar_type> rmm_input_cpu(COALESCED_DIMENSION(group_m), group_m+DENSITY_BLOCK_SIZE);
  get_rmm_input(rmm_input_cpu); //Achica la matriz densidad a la version reducida del grupo

  for (uint i=0; i<(group_m+DENSITY_BLOCK_SIZE); i++)
  {
    for(uint j=0; j<COALESCED_DIMENSION(group_m); j++)
    {
      if((i>=group_m) || (j>=group_m) || (j > i))
      {
        rmm_input_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=0.0f;
      }
    }
  }


  /*
   **********************************************************************
   * Pasando RDM (rmm) a texturas
   **********************************************************************
   */

  //Comentado porque ahora vamos a hacer esto a mano por la textura
  // TODO: pasarlo a un metodo dentro de matrix.cpp
  //rmm_input_gpu = rmm_input_cpu; //Aca copia de CPU a GPU

  hipArray* cuArray;
  hipMallocArray(&cuArray, &rmm_input_gpu_tex.channelDesc, rmm_input_cpu.width,rmm_input_cpu.height);
#if FULL_DOUBLE
  hipMemcpyToArray(cuArray, 0, 0,rmm_input_cpu.data,sizeof(int2)*rmm_input_cpu.width*rmm_input_cpu.height, hipMemcpyHostToDevice);
#else
  hipMemcpyToArray(cuArray, 0, 0,rmm_input_cpu.data,sizeof(float)*rmm_input_cpu.width*rmm_input_cpu.height, hipMemcpyHostToDevice);
#endif
  hipBindTextureToArray(rmm_input_gpu_tex, cuArray);

  rmm_input_gpu_tex.normalized = false;

  if (compute_energy) {
    CudaMatrix<scalar_type> energy_gpu(number_of_points);
#define compute_parameters \
    energy_gpu.data,factors_gpu.data,point_weights_gpu.data,number_of_points,function_values_transposed_gpu.data,gradient_values_transposed_gpu.data,hessian_values_transposed_gpu.data,group_m,partial_densities_gpu.data,dxyz_gpu.data,dd1_gpu.data,dd2_gpu.data
#define accumulate_parameters \
    energy_gpu.data,factors_gpu.data,point_weights_gpu.data,number_of_points,block_height,partial_densities_gpu.data,dxyz_gpu.data,dd1_gpu.data,dd2_gpu.data
    if (compute_forces || compute_rmm) {
      if (lda)
      {
          gpu_compute_density<scalar_type, true, true, true><<<threadGrid, threadBlock>>>(compute_parameters);
          gpu_accumulate_point<scalar_type, true, true, true><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
      }
      else
      {
          gpu_compute_density<scalar_type, true, true, false><<<threadGrid, threadBlock>>>(compute_parameters);
          gpu_accumulate_point<scalar_type, true, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
      }
    }
    else {
      if (lda)
      {
          gpu_compute_density<scalar_type, true, false, true><<<threadGrid, threadBlock>>>(compute_parameters);
          gpu_accumulate_point<scalar_type, true, false, true><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
      }
      else
      {
          gpu_compute_density<scalar_type, true, false, false><<<threadGrid, threadBlock>>>(compute_parameters);
          gpu_accumulate_point<scalar_type, true, false, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
      }
    }
    cudaAssertNoError("compute_density");

    HostMatrix<scalar_type> energy_cpu(energy_gpu);
        for (uint i = 0; i < number_of_points; i++) {
        energy += energy_cpu(i);
    } // TODO: hacer con un kernel?
  }
  else {
#undef compute_parameters
#undef accumulate_parameters

#define compute_parameters \
    NULL,factors_gpu.data,point_weights_gpu.data,number_of_points,function_values_transposed_gpu.data,gradient_values_transposed_gpu.data,hessian_values_transposed_gpu.data,group_m,partial_densities_gpu.data,dxyz_gpu.data,dd1_gpu.data,dd2_gpu.data
#define accumulate_parameters \
    NULL,factors_gpu.data,point_weights_gpu.data,number_of_points,block_height,partial_densities_gpu.data,dxyz_gpu.data,dd1_gpu.data,dd2_gpu.data
    if (lda)
    {
        gpu_compute_density<scalar_type, false, true, true><<<threadGrid, threadBlock>>>(compute_parameters);
        gpu_accumulate_point<scalar_type, false, true, true><<<threadGrid_accumulate, threadBlock_accumulate>>>(accumulate_parameters);
    }
    else
    {
        gpu_compute_density<scalar_type, false, true, false><<<threadGrid, threadBlock>>>(compute_parameters);
        gpu_accumulate_point<scalar_type, false, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>>(accumulate_parameters);
    }
    cudaAssertNoError("compute_density");
  }
#undef compute_parameters
#undef accumulate_parameters

  function_values_transposed_gpu.deallocate();
  gradient_values_transposed_gpu.deallocate();
  hessian_values_transposed_gpu.deallocate();

  timers.density.pause_and_sync();

//************ Repongo los valores que puse a cero antes, para las fuerzas son necesarios (o por lo mens utiles)
  for (uint i=0; i<(group_m); i++)
  {
    for(uint j=0; j<(group_m); j++)
    {
      if((i>=group_m) || (j>=group_m) || (j > i))
      {
        rmm_input_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=rmm_input_cpu.data[COALESCED_DIMENSION(group_m)*j+i] ;
      }
    }
  }
#if FULL_DOUBLE
  hipMemcpyToArray(cuArray, 0, 0,rmm_input_cpu.data,sizeof(int2)*rmm_input_cpu.width*rmm_input_cpu.height, hipMemcpyHostToDevice);
#else
  hipMemcpyToArray(cuArray, 0, 0,rmm_input_cpu.data,sizeof(float)*rmm_input_cpu.width*rmm_input_cpu.height, hipMemcpyHostToDevice);
#endif

//**********************************************

   dim3 threads;
  /* compute forces */
  if (compute_forces) {
    timers.density_derivs.start_and_sync();
    threads = dim3(number_of_points);
    threadBlock = dim3(DENSITY_DERIV_BLOCK_SIZE);
    threadGrid = divUp(threads, threadBlock);

    CudaMatrix<vec_type4> dd_gpu(COALESCED_DIMENSION(number_of_points), total_nucleii()); dd_gpu.zero();
    CudaMatrixUInt nuc_gpu(func2local_nuc);  // TODO: esto en realidad se podria guardar una sola vez durante su construccion

    gpu_compute_density_derivs<<<threadGrid, threadBlock>>>(function_values.data, gradient_values.data, nuc_gpu.data, dd_gpu.data, number_of_points, group_m, total_nucleii());
    cudaAssertNoError("density_derivs");
    timers.density_derivs.pause_and_sync();

    timers.forces.start_and_sync();
    CudaMatrix<vec_type4> forces_gpu(total_nucleii());

    threads = dim3(total_nucleii());
    threadBlock = dim3(FORCE_BLOCK_SIZE);
    threadGrid = divUp(threads, threadBlock);
    gpu_compute_forces<<<threadGrid, threadBlock>>>(number_of_points, factors_gpu.data, dd_gpu.data, forces_gpu.data, total_nucleii());
    cudaAssertNoError("forces");

    HostMatrix<vec_type4> forces_cpu(forces_gpu);

    for (uint i = 0; i < total_nucleii(); ++i) {
      vec_type4 atom_force = forces_cpu(i);
      uint global_nuc = local2global_nuc[i];
      fort_forces(global_nuc, 0) += atom_force.x;
      fort_forces(global_nuc, 1) += atom_force.y;
      fort_forces(global_nuc, 2) += atom_force.z;
    }
    timers.forces.pause_and_sync();
  }

  timers.rmm.start_and_sync();
  /* compute RMM */
  if (compute_rmm) {
    threadBlock = dim3(RMM_BLOCK_SIZE_XY, RMM_BLOCK_SIZE_XY);
    uint blocksPerRow = divUp(group_m, RMM_BLOCK_SIZE_XY);
    // Only use enough blocks for lower triangle
    threadGrid = dim3(blocksPerRow*(blocksPerRow+1)/2);

    CudaMatrix<scalar_type> rmm_output_gpu(COALESCED_DIMENSION(group_m), group_m);
    // For calls with a single block (pretty common with cubes) don't bother doing the arithmetic to get block position in the matrix
    if (blocksPerRow > 1) {
        gpu_update_rmm<scalar_type,true><<<threadGrid, threadBlock>>>(factors_gpu.data, number_of_points, rmm_output_gpu.data, function_values.data, group_m);
    } else {
        gpu_update_rmm<scalar_type,false><<<threadGrid, threadBlock>>>(factors_gpu.data, number_of_points, rmm_output_gpu.data, function_values.data, group_m);
    }
    cudaAssertNoError("update_rmm");

    /*** Contribute this RMM to the total RMM ***/
    HostMatrix<scalar_type> rmm_output_cpu(rmm_output_gpu);
    add_rmm_output(rmm_output_cpu);
  }
  timers.rmm.pause_and_sync();

  /* clear functions */
  if(!(this->inGlobal)) {
    function_values.deallocate();
    gradient_values.deallocate();
    hessian_values.deallocate();
  }
  //Deshago el bind de textura de rmm
  hipUnbindTexture(rmm_input_gpu_tex); //Enroque el Unbind con el Free, asi parece mas logico. Nano
  hipFreeArray(cuArray);
}

//======================
// OPENSHELL
//======================
template<class scalar_type>
void PointGroup<scalar_type>::solve_opened(Timers& timers, bool compute_rmm, bool lda, bool compute_forces, bool compute_energy, double& energy, double& energy_i, double& energy_c, double& energy_c1, double& energy_c2, double* fort_forces_ptr){
//  if(open){
//    cout<<"!!!!!!"<<endl;
//    cout<<"ENTRANDO A SOLVE !!!!!!"<<endl;
//    cout<<"!!!!!!"<<endl;
//  }
  //uint max_used_memory = 0;

  /*** Computo sobre cada cubo ****/
  CudaMatrix<scalar_type> point_weights_gpu;
  FortranMatrix<double> fort_forces(fort_forces_ptr, fortran_vars.atoms, 3, fortran_vars.max_atoms);

  /** Compute this group's functions **/
  timers.functions.start_and_sync();
  compute_functions(compute_forces, !lda); //<<<<==========
  timers.functions.pause_and_sync();

  uint group_m = total_functions();

  timers.density.start_and_sync();
  /** Load points from group **/
  HostMatrix<scalar_type> point_weights_cpu(number_of_points, 1);

  uint i = 0;
  for (list<Point>::const_iterator p = points.begin(); p != points.end(); ++p, ++i) {
    point_weights_cpu(i) = p->weight;
  }
  point_weights_gpu = point_weights_cpu;

//<<===========================>>//
  dim3 threadBlock, threadGrid;
  /* compute density/factors */
 /** New code (por funciones) **/

  const int block_height= divUp(group_m,2*DENSITY_BLOCK_SIZE);

  threadBlock = dim3(DENSITY_BLOCK_SIZE,1,1); // Hay que asegurarse que la cantidad de funciones este en rango
  threadGrid = dim3(number_of_points,block_height,1);

  CudaMatrix<scalar_type> factors_a_gpu;
  CudaMatrix<scalar_type> factors_b_gpu;

/*
  CudaMatrix<scalar_type> partial_densities_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dxyz_gpu; gradiente
  CudaMatrix<vec_type<scalar_type,4> > dd1_gpu;  hessiano ii
  CudaMatrix<vec_type<scalar_type,4> > dd2_gpu;  hessiano ij
*/

  CudaMatrix<scalar_type> partial_densities_a_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dxyz_a_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd1_a_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd2_a_gpu;

  CudaMatrix<scalar_type> partial_densities_b_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dxyz_b_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd1_b_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd2_b_gpu;

  /*
   **********************************************************************
   * Transposiciones de matrices para la coalescencia mejorada en density
   **********************************************************************
   */

  CudaMatrix<scalar_type>              function_values_transposed_gpu;
  CudaMatrix<vec_type<scalar_type,4> > gradient_values_transposed_gpu;
  CudaMatrix<vec_type<scalar_type,4> > hessian_values_transposed_gpu;

  int transposed_width = COALESCED_DIMENSION(number_of_points);

  function_values_transposed_gpu.resize(group_m, COALESCED_DIMENSION(number_of_points));
  if (fortran_vars.do_forces || fortran_vars.gga)
      gradient_values_transposed_gpu.resize( group_m,COALESCED_DIMENSION(number_of_points));
  if (fortran_vars.gga)
      hessian_values_transposed_gpu.resize((group_m) * 2, COALESCED_DIMENSION(number_of_points));

  #define BLOCK_DIM 16
  dim3 transpose_grid(transposed_width / BLOCK_DIM, divUp((group_m),BLOCK_DIM));
  dim3 transpose_threads(BLOCK_DIM, BLOCK_DIM, 1);

  transpose<<<transpose_grid, transpose_threads>>> (function_values_transposed_gpu.data, function_values.data,  COALESCED_DIMENSION(number_of_points),group_m   );
  if (fortran_vars.do_forces || fortran_vars.gga)
      transpose_vec<<<transpose_grid, transpose_threads>>> (gradient_values_transposed_gpu.data, gradient_values.data, COALESCED_DIMENSION(number_of_points), group_m );
  transpose_grid=dim3(transposed_width / BLOCK_DIM, divUp((group_m)*2, BLOCK_DIM), 1);
  if (fortran_vars.gga)
      transpose_vec<<<transpose_grid, transpose_threads>>> (hessian_values_transposed_gpu.data, hessian_values.data, COALESCED_DIMENSION(number_of_points), (group_m)*2);

//=====

//  partial_densities_gpu.resize(COALESCED_DIMENSION(number_of_points), block_height);
//  dxyz_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height);
//  dd1_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height );
//  dd2_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height );

  partial_densities_a_gpu.resize(COALESCED_DIMENSION(number_of_points), block_height);
  dxyz_a_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height);
  dd1_a_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height );
  dd2_a_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height );

  partial_densities_b_gpu.resize(COALESCED_DIMENSION(number_of_points), block_height);
  dxyz_b_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height);
  dd1_b_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height );
  dd2_b_gpu.resize(COALESCED_DIMENSION(number_of_points),block_height );

  const dim3 threadGrid_accumulate(divUp(number_of_points,DENSITY_ACCUM_BLOCK_SIZE),1,1);
  const dim3 threadBlock_accumulate(DENSITY_ACCUM_BLOCK_SIZE,1,1);


  if (compute_rmm || compute_forces){
  	factors_a_gpu.resize(number_of_points);
  	factors_b_gpu.resize(number_of_points);
  }
//
//  HostMatrix<scalar_type> rmm_input_cpu(COALESCED_DIMENSION(group_m), group_m+DENSITY_BLOCK_SIZE);
//  get_rmm_input(rmm_input_cpu); //Achica la matriz densidad a la version reducida del grupo
//
//==============================================
// NUEVO ....
  HostMatrix<scalar_type> rmm_input_a_cpu(COALESCED_DIMENSION(group_m), group_m+DENSITY_BLOCK_SIZE);
  HostMatrix<scalar_type> rmm_input_b_cpu(COALESCED_DIMENSION(group_m), group_m+DENSITY_BLOCK_SIZE);
  get_rmm_input(rmm_input_a_cpu,rmm_input_b_cpu); //Achica las matrices densidad (Up,Down) a la version reducida del grupo
//===============================================

  for (uint i=0; i<(group_m+DENSITY_BLOCK_SIZE); i++)
  {
    for(uint j=0; j<COALESCED_DIMENSION(group_m); j++)
    {
      if((i>=group_m) || (j>=group_m) || (j > i))
      {
        rmm_input_a_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=0.0f;
        rmm_input_b_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=0.0f;
      }
    }
  }

  /*
   **********************************************************************
   * Pasando RDM (rmm) a texturas/
   **********************************************************************
   */

  //Comentado porque ahora vamos a hacer esto a mano por la textura
  // TODO: pasarlo a un metodo dentro de matrix.cpp
  //rmm_input_gpu = rmm_input_cpu; //Aca copia de CPU a GPU

  hipArray* cuArray1;
  hipArray* cuArray2;
  hipMallocArray(&cuArray1, &rmm_input_gpu_tex.channelDesc, rmm_input_a_cpu.width,rmm_input_a_cpu.height);
  hipMallocArray(&cuArray2, &rmm_input_gpu_tex2.channelDesc, rmm_input_b_cpu.width,rmm_input_b_cpu.height);
#if FULL_DOUBLE
  hipMemcpyToArray(cuArray1, 0, 0,rmm_input_a_cpu.data,sizeof(int2)*rmm_input_a_cpu.width*rmm_input_a_cpu.height, hipMemcpyHostToDevice);
  hipMemcpyToArray(cuArray2, 0, 0,rmm_input_b_cpu.data,sizeof(int2)*rmm_input_b_cpu.width*rmm_input_b_cpu.height, hipMemcpyHostToDevice);
#else
  hipMemcpyToArray(cuArray1, 0, 0,rmm_input_a_cpu.data,sizeof(float)*rmm_input_a_cpu.width*rmm_input_a_cpu.height, hipMemcpyHostToDevice);
  hipMemcpyToArray(cuArray2, 0, 0,rmm_input_b_cpu.data,sizeof(float)*rmm_input_b_cpu.width*rmm_input_b_cpu.height, hipMemcpyHostToDevice);
#endif
  hipBindTextureToArray(rmm_input_gpu_tex, cuArray1);
  hipBindTextureToArray(rmm_input_gpu_tex2, cuArray2);

/*
  void* devPtr;
  size_t pPitch;
  size_t row_width = rmm_input_cpu.width*sizeof(float);
  size_t row_height = rmm_input_cpu.height;
  size_t offset;
  hipMallocPitch(&devPtr, &pPitch, row_width ,row_height);
  hipMemcpy2D(devPtr, pPitch, rmm_input_cpu.data, row_width, row_width, row_height,hipMemcpyHostToDevice);
  hipBindTexture2D(&offset, rmm_input_gpu_tex, devPtr, rmm_input_gpu_tex.channelDesc, rmm_input_cpu.width, row_height, pPitch);
*/
  rmm_input_gpu_tex.normalized = false;
  rmm_input_gpu_tex2.normalized = false;

  if (compute_energy) {

      CudaMatrix<scalar_type> energy_gpu(number_of_points);
      CudaMatrix<scalar_type> energy_i_gpu(number_of_points);
      CudaMatrix<scalar_type> energy_c_gpu(number_of_points);
      CudaMatrix<scalar_type> energy_c1_gpu(number_of_points);
      CudaMatrix<scalar_type> energy_c2_gpu(number_of_points);


      if (compute_forces || compute_rmm){
//         if (lda) {
//       template<class scalar_type, bool compute_energy, bool compute_factor, bool lda>
//             gpu_compute_density<scalar_type, true, true, true><<<threadGrid, threadBlock>>>(energy_gpu.data, factors_gpu.data, point_weights_gpu.data, number_of_points,  function_values_transposed_gpu.data, gradient_values_transposed_gpu.data, hessian_values_transposed_gpu.data, group_m, partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data);
//             gpu_accumulate_point<scalar_type, true, true, true><<<threadGrid_accumulate, threadBlock_accumulate>>> (energy_gpu.data, factors_gpu.data, point_weights_gpu.data,number_of_points,block_height, partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data);
//         }
//         else{

	     	//cout<<"ENTRANDO a gpu_compute_density_opened..."<<endl;
             	gpu_compute_density_opened<scalar_type, true, true, false><<<threadGrid, threadBlock>>>(
                                        point_weights_gpu.data,number_of_points, function_values_transposed_gpu.data,
 					gradient_values_transposed_gpu.data,hessian_values_transposed_gpu.data, group_m,
                                        partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
                                        partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);

		//cout<<"ENTRANDO a gpu_accumulate_point_open..."<<endl;
             	gpu_accumulate_point_open<scalar_type, true, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
                                  energy_gpu.data,energy_i_gpu.data,energy_c_gpu.data,energy_c1_gpu.data,energy_c2_gpu.data,
                                  factors_a_gpu.data, factors_b_gpu.data, point_weights_gpu.data,number_of_points,block_height,
                                  partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
				  partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
//         }
      }
      else{
//          if(lda){
//              gpu_compute_density<scalar_type, true, false, true><<<threadGrid, threadBlock>>>(energy_gpu.data, factors_gpu.data, point_weights_gpu.data, number_of_points, function_values_transposed_gpu.data, gradient_values_transposed_gpu.data, hessian_values_transposed_gpu.data, group_m, partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data);
//              gpu_accumulate_point<scalar_type, true, false, true><<<threadGrid_accumulate, threadBlock_accumulate>>> (energy_gpu.data, factors_gpu.data, point_weights_gpu.data,number_of_points,block_height, partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data);
//          }
//          else{
              gpu_compute_density_opened<scalar_type, true, false, false><<<threadGrid, threadBlock>>>(
                                         point_weights_gpu.data,number_of_points, function_values_transposed_gpu.data,
					 gradient_values_transposed_gpu.data,hessian_values_transposed_gpu.data, group_m,
                                         partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
                                         partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
              gpu_accumulate_point_open<scalar_type, true, false, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
                                   energy_gpu.data, energy_i_gpu.data,energy_c_gpu.data,energy_c1_gpu.data,energy_c2_gpu.data,
                                   factors_a_gpu.data, factors_b_gpu.data, point_weights_gpu.data,number_of_points,block_height,
                                   partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
                                   partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
//          }
      }
      cudaAssertNoError("compute_density");

      HostMatrix<scalar_type> energy_cpu(energy_gpu);
      HostMatrix<scalar_type> energy_i_cpu(energy_i_gpu);
      HostMatrix<scalar_type> energy_c_cpu(energy_c_gpu);
      HostMatrix<scalar_type> energy_c1_cpu(energy_c1_gpu);
      HostMatrix<scalar_type> energy_c2_cpu(energy_c2_gpu);
      for (uint i = 0; i < number_of_points; i++) {
          energy    += energy_cpu(i);
          energy_i  += energy_i_cpu(i);
          energy_c  += energy_c_cpu(i);
          energy_c1 += energy_c1_cpu(i);
          energy_c2 += energy_c2_cpu(i);
      } // TODO: hacer con un kernel?
  }
  else{
//      if (lda){
//          gpu_compute_density<scalar_type, false, true, true><<<threadGrid, threadBlock>>>(NULL, factors_gpu.data, point_weights_gpu.data, number_of_points, function_values_transposed_gpu.data, gradient_values_transposed_gpu.data, hessian_values_transposed_gpu.data, group_m, partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data);
//          gpu_accumulate_point<scalar_type, false, true, true><<<threadGrid_accumulate, threadBlock_accumulate>>> (NULL, factors_gpu.data, point_weights_gpu.data,number_of_points,block_height, partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data);
//      }
//      else{
          gpu_compute_density_opened<scalar_type, false, true, false><<<threadGrid, threadBlock>>>(
                                     point_weights_gpu.data, number_of_points, function_values_transposed_gpu.data,
    				     gradient_values_transposed_gpu.data,hessian_values_transposed_gpu.data, group_m,
                                     partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
                                     partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
          gpu_accumulate_point_open<scalar_type, false, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
                               NULL,NULL,NULL,NULL,NULL,
                               factors_a_gpu.data, factors_b_gpu.data, point_weights_gpu.data,number_of_points,block_height,
                               partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
                               partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
//      }
      cudaAssertNoError("compute_density");
  }

  function_values_transposed_gpu.deallocate();
  gradient_values_transposed_gpu.deallocate();
  hessian_values_transposed_gpu.deallocate();

  timers.density.pause_and_sync();

//************ Repongo los valores que puse a cero antes, para las fuerzas son necesarios (o por lo mens utiles)
  for (uint i=0; i<(group_m); i++){
    for(uint j=0; j<(group_m); j++){
      if((i>=group_m) || (j>=group_m) || (j > i)){
        rmm_input_a_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=rmm_input_a_cpu.data[COALESCED_DIMENSION(group_m)*j+i] ;
        rmm_input_b_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=rmm_input_b_cpu.data[COALESCED_DIMENSION(group_m)*j+i] ;
      }
    }
  }
#if FULL_DOUBLE
  hipMemcpyToArray(cuArray1, 0, 0,rmm_input_a_cpu.data,sizeof(int2)*rmm_input_a_cpu.width*rmm_input_a_cpu.height, hipMemcpyHostToDevice);
  hipMemcpyToArray(cuArray2, 0, 0,rmm_input_b_cpu.data,sizeof(int2)*rmm_input_b_cpu.width*rmm_input_b_cpu.height, hipMemcpyHostToDevice);
#else
  hipMemcpyToArray(cuArray1, 0, 0,rmm_input_a_cpu.data,sizeof(float)*rmm_input_a_cpu.width*rmm_input_a_cpu.height, hipMemcpyHostToDevice);
  hipMemcpyToArray(cuArray2, 0, 0,rmm_input_b_cpu.data,sizeof(float)*rmm_input_b_cpu.width*rmm_input_b_cpu.height, hipMemcpyHostToDevice);
#endif

//**********************************************

   dim3 threads;
  /* compute forces */

  if (compute_forces) {
  	timers.density_derivs.start_and_sync();
    	    threads = dim3(number_of_points);
    	    threadBlock = dim3(DENSITY_DERIV_BLOCK_SIZE);
    	    threadGrid = divUp(threads, threadBlock);

    	    CudaMatrix<vec_type4> dd_gpu_a(COALESCED_DIMENSION(number_of_points), total_nucleii());
    	    CudaMatrix<vec_type4> dd_gpu_b(COALESCED_DIMENSION(number_of_points), total_nucleii());
            dd_gpu_a.zero();
            dd_gpu_b.zero();
    	    CudaMatrixUInt nuc_gpu(func2local_nuc);  // TODO: esto en realidad se podria guardar una sola vez durante su construccion

	    // Kernel
    	    gpu_compute_density_derivs_open<<<threadGrid, threadBlock>>>(function_values.data, gradient_values.data, nuc_gpu.data, dd_gpu_a.data, dd_gpu_b.data, number_of_points, group_m, total_nucleii());

    	    cudaAssertNoError("density_derivs");
    	timers.density_derivs.pause_and_sync();

    	timers.forces.start_and_sync();
    	    CudaMatrix<vec_type4> forces_gpu_a(total_nucleii());
    	    CudaMatrix<vec_type4> forces_gpu_b(total_nucleii());

      	    threads = dim3(total_nucleii());
    	    threadBlock = dim3(FORCE_BLOCK_SIZE);
    	    threadGrid = divUp(threads, threadBlock);
    	    // Kernel
	    gpu_compute_forces<<<threadGrid, threadBlock>>>(number_of_points, factors_a_gpu.data, dd_gpu_a.data, forces_gpu_a.data, total_nucleii());
	    gpu_compute_forces<<<threadGrid, threadBlock>>>(number_of_points, factors_b_gpu.data, dd_gpu_b.data, forces_gpu_b.data, total_nucleii());

//            gpu_compute_forces_open<<<threadGrid, threadBlock>>>(number_of_points, factors_a_gpu.data, factors_b_gpu.data, dd_gpu_a.data,dd_gpu_b.data, forces_gpu_a.data, forces_gpu_b.data, total_nucleii());

    	    cudaAssertNoError("forces");

    	    HostMatrix<vec_type4> forces_cpu_a(forces_gpu_a);
    	    HostMatrix<vec_type4> forces_cpu_b(forces_gpu_b);

    	    for (uint i = 0; i < total_nucleii(); ++i) {
      	    	vec_type4 atom_force_a = forces_cpu_a(i);
      	    	vec_type4 atom_force_b = forces_cpu_b(i);
            	uint global_nuc = local2global_nuc[i];

                fort_forces(global_nuc, 0)=fort_forces(global_nuc, 0) + atom_force_a.x + atom_force_b.x;
		fort_forces(global_nuc, 1)=fort_forces(global_nuc, 1) + atom_force_a.y + atom_force_b.y;
		fort_forces(global_nuc, 2)=fort_forces(global_nuc, 2) + atom_force_a.z + atom_force_b.z;

//                cout<<"force.x="<<atom_force_a.x+atom_force_b.x<<"force.y="<<atom_force_a.y+atom_force_b.y<<"force.z="<<atom_force_a.z+atom_force_b.z<<endl;
            }

    	timers.forces.pause_and_sync();
  }

  /* compute RMM */
  timers.rmm.start_and_sync();
  if (compute_rmm) {
	//cout<<"EMPEZANDO GPU_UPDATE_RMM"<<endl;
	//threads = dim3(group_m, group_m);
    	threadBlock = dim3(RMM_BLOCK_SIZE_XY, RMM_BLOCK_SIZE_XY);
    	//threadGrid = divUp(threads, threadBlock);
        uint blocksPerRow = divUp(group_m, RMM_BLOCK_SIZE_XY);
        // Only use enough blocks for lower triangle
        threadGrid = dim3(blocksPerRow*(blocksPerRow+1)/2);

    	CudaMatrix<scalar_type> rmm_output_a_gpu(COALESCED_DIMENSION(group_m), group_m);
    	CudaMatrix<scalar_type> rmm_output_b_gpu(COALESCED_DIMENSION(group_m), group_m);
    	// Kernel
//	cout<<"alpha"<<endl;
        // For calls with a single block (pretty common with cubes) don't bother doing the arithmetic to get block position in the matrix
        if (blocksPerRow > 1) {
	    gpu_update_rmm<scalar_type,true><<<threadGrid, threadBlock>>>(factors_a_gpu.data, number_of_points, rmm_output_a_gpu.data, function_values.data, group_m);
//	cout<<endl;
//        cout<<"beta"<<endl;
            gpu_update_rmm<scalar_type,true><<<threadGrid, threadBlock>>>(factors_b_gpu.data, number_of_points, rmm_output_b_gpu.data, function_values.data, group_m);
        } else {
	    gpu_update_rmm<scalar_type,false><<<threadGrid, threadBlock>>>(factors_a_gpu.data, number_of_points, rmm_output_a_gpu.data, function_values.data, group_m);
            gpu_update_rmm<scalar_type,false><<<threadGrid, threadBlock>>>(factors_b_gpu.data, number_of_points, rmm_output_b_gpu.data, function_values.data, group_m);
        }
    	//cout<<endl;
        cudaAssertNoError("update_rmm");

    	/*** Contribute this RMM to the total RMM ***/
    	HostMatrix<scalar_type> rmm_output_a_cpu(rmm_output_a_gpu);
    	HostMatrix<scalar_type> rmm_output_b_cpu(rmm_output_b_gpu);
    	//add_rmm_open_output(rmm_output_a_cpu,rmm_output_b_cpu);
    	add_rmm_output_a(rmm_output_a_cpu);
    	add_rmm_output_b(rmm_output_b_cpu);

  }
  timers.rmm.pause_and_sync();

  /* clear functions */
  if(!(this->inGlobal))
  {
        function_values.deallocate();
        gradient_values.deallocate();
        hessian_values.deallocate();
  }

  //Deshago el bind de textura de rmm
  hipUnbindTexture(rmm_input_gpu_tex); //Enroque el Unbind con el Free, asi parece mas logico. Nano
  hipUnbindTexture(rmm_input_gpu_tex2); //Enroque el Unbind con el Free, asi parece mas logico. Nano
  hipFreeArray(cuArray1);
  hipFreeArray(cuArray2);
  hipFree(cuArray1);
  hipFree(cuArray2);

  //uint free_memory, total_memory;
  //cudaGetMemoryInfo(free_memory, total_memory);
  //cout << "Maximum used memory: " << (double)max_used_memory / (1024 * 1024) << "MB (" << ((double)max_used_memory / total_memory) * 100.0 << "%)" << endl;
  //cudaPrintMemoryInfo();
}

/*******************************
 * Cube Functions
 *******************************/

template<class scalar_type>
void PointGroup<scalar_type>::compute_functions(bool forces, bool gga)
{
  if(this->inGlobal) //Ya las tengo en memoria? entonces salgo porque ya estan las 3 calculadas
    return;

  if(0 == globalMemoryPool::tryAlloc(this->size_in_gpu())) //1 si hubo error, 0 si pude reservar la memoria
    this->inGlobal=true;
  CudaMatrix<vec_type4> points_position_gpu;
  CudaMatrix<vec_type2> factor_ac_gpu;
  CudaMatrixUInt nuc_gpu;
  CudaMatrixUInt contractions_gpu;

  /** Load points from group **/
  {
    HostMatrix<vec_type4> points_position_cpu(number_of_points, 1);
    uint i = 0;
    for (list<Point>::const_iterator p = points.begin(); p != points.end(); ++p, ++i) {
      points_position_cpu(i) = vec_type4(p->position.x, p->position.y, p->position.z, 0);
    }
    points_position_gpu = points_position_cpu;
  }
  /* Load group functions */
  uint group_m = s_functions + p_functions * 3 + d_functions * 6;
  uint4 group_functions = make_uint4(s_functions, p_functions, d_functions, group_m);
  HostMatrix<vec_type2> factor_ac_cpu(COALESCED_DIMENSION(group_m), MAX_CONTRACTIONS);
  HostMatrixUInt nuc_cpu(group_m, 1), contractions_cpu(group_m, 1);

  // TODO: hacer que functions.h itere por total_small_functions()... asi puedo hacer que
  // func2global_nuc sea de tamaño total_functions() y directamente copio esa matriz aca y en otros lados

  uint ii = 0;
  for (uint i = 0; i < total_functions_simple(); ++i) {
    uint inc = small_function_type(i);

    uint func = local2global_func[i];
    uint this_nuc = func2global_nuc(i);
    uint this_cont = fortran_vars.contractions(func);

    for (uint j = 0; j < inc; j++) {
      nuc_cpu(ii) = this_nuc;
      contractions_cpu(ii) = this_cont;
      for (unsigned int k = 0; k < this_cont; k++)
        factor_ac_cpu(ii, k) = vec_type2(fortran_vars.a_values(func, k), fortran_vars.c_values(func, k));
      ii++;
    }
  }
  factor_ac_gpu = factor_ac_cpu;
  nuc_gpu = nuc_cpu;
  contractions_gpu = contractions_cpu;

  /** Compute Functions **/

  function_values.resize(COALESCED_DIMENSION(number_of_points), group_functions.w);
  if (fortran_vars.do_forces || fortran_vars.gga)
      gradient_values.resize(COALESCED_DIMENSION(number_of_points), group_functions.w);
  if (fortran_vars.gga)
      hessian_values.resize(COALESCED_DIMENSION(number_of_points), (group_functions.w) * 2);

  dim3 threads(number_of_points);
  dim3 threadBlock(FUNCTIONS_BLOCK_SIZE);
  dim3 threadGrid = divUp(threads, threadBlock);

 // cout << "points: " << threads.x << " " << threadGrid.x << " " << threadBlock.x << endl;
#define compute_functions_parameters \
  points_position_gpu.data,number_of_points,contractions_gpu.data,factor_ac_gpu.data,nuc_gpu.data,function_values.data,gradient_values.data,hessian_values.data,group_functions
  if (forces) {
    if (gga)
      gpu_compute_functions<scalar_type, true, true><<<threadGrid, threadBlock>>>(compute_functions_parameters);
    else
      gpu_compute_functions<scalar_type, true, false><<<threadGrid, threadBlock>>>(compute_functions_parameters);
  }
  else {
    if (gga)
      gpu_compute_functions<scalar_type, false, true><<<threadGrid, threadBlock>>>(compute_functions_parameters);
    else
      gpu_compute_functions<scalar_type, false, false><<<threadGrid, threadBlock>>>(compute_functions_parameters);
  }

  cudaAssertNoError("compute_functions");
}

/*******************************
 * Cube Weights
 *******************************/
template<class scalar_type>
void PointGroup<scalar_type>::compute_weights(void)
{
  CudaMatrix<vec_type4> point_positions_gpu;
  CudaMatrix<vec_type4> atom_position_rm_gpu;
  {
    HostMatrix<vec_type4> points_positions_cpu(number_of_points, 1);
		uint i = 0;
		for (list<Point>::const_iterator p = points.begin(); p != points.end(); ++p, ++i) {
			points_positions_cpu(i) = vec_type4(p->position.x, p->position.y, p->position.z, p->atom);
		}
    point_positions_gpu = points_positions_cpu;

    HostMatrix<vec_type4> atom_position_rm_cpu(fortran_vars.atoms, 1);
    for (uint i = 0; i < fortran_vars.atoms; i++) {
      double3 atom_pos = fortran_vars.atom_positions(i);
      atom_position_rm_cpu(i) = vec_type4(atom_pos.x, atom_pos.y, atom_pos.z, fortran_vars.rm(i));
    }
    atom_position_rm_gpu = atom_position_rm_cpu;
  }

  CudaMatrixUInt nucleii_gpu(local2global_nuc);

  CudaMatrix<scalar_type> weights_gpu(number_of_points);
  dim3 threads(number_of_points);
  dim3 blockSize(WEIGHT_BLOCK_SIZE);
  dim3 gridSize = divUp(threads, blockSize);
  gpu_compute_weights<scalar_type><<<gridSize,blockSize>>>(
      number_of_points, point_positions_gpu.data, atom_position_rm_gpu.data, weights_gpu.data, nucleii_gpu.data, total_nucleii());
  cudaAssertNoError("compute_weights");

  #if REMOVE_ZEROS
  std::list<Point> nonzero_points;
  uint nonzero_number_of_points = 0;
  #endif

  uint ceros = 0;

  HostMatrix<scalar_type> weights_cpu(weights_gpu);
  uint i = 0;
  for (list<Point>::iterator p = points.begin(); p != points.end(); ++p, ++i) {
    p->weight *= weights_cpu(i);

    if (p->weight == 0.0) {
      ceros++;
    }
    #if REMOVE_ZEROS
    else {
      nonzero_points.push_back(*p);
      nonzero_number_of_points++;
    }
    #endif
  }

  //cout << "ceros: " << ceros << "/" << group.number_of_points << " (" << (ceros / (double)group.number_of_points) * 100 << "%)" << endl;

  #if REMOVE_ZEROS
  points = nonzero_points;
  number_of_points = nonzero_number_of_points;
  #endif
}

template class PointGroup<double>;
template class PointGroup<float>;

template <class scalar_type> void get_qmmm_forces(double* qm_forces, double* mm_forces)
{
  uint i,j,ni,nj;
  uint i_orbitals, j_orbitals;
  uint nuc_i,nuc_j;
  vec_type<double,3> A,B,AmB;
  double ai,aj;
  double dsq,ksi,zeta;
  uint num_terms=0, total_num_terms = 0;
  //std::vector<uint> local2func1,local2func2;
  std::vector<scalar_type> a_values1,a_values2;
  std::vector<scalar_type> cc_values;
  std::vector<scalar_type> dens_values;
  std::vector<uint> nuclei1, nuclei2;

  // function i, center A
  i = 0;
  while (i < fortran_vars.s_funcs) {//m) {
    nuc_i = fortran_vars.nucleii(i) - 1;
    A = fortran_vars.atom_positions(nuc_i);
    if (i < fortran_vars.s_funcs) {
      i_orbitals = 1;
    } else if (i < fortran_vars.s_funcs + fortran_vars.p_funcs*3) {
      i_orbitals = 3;
    } else {
      i_orbitals = 6;
    }
    // function j, center B
    j = 0;
    while (j <= i) {
      nuc_j = fortran_vars.nucleii(j) - 1;
      B = fortran_vars.atom_positions(nuc_j);
      if (j < fortran_vars.s_funcs) {
        j_orbitals = 1;
      } else if (j < fortran_vars.s_funcs + fortran_vars.p_funcs*3) {
        j_orbitals = 3;
      } else {
        j_orbitals = 6;
      }
      AmB = A - B;
      dsq = length2(AmB);
      uint dens_ind = i + (2*fortran_vars.m-(j+1))*j/2;

      for (ni = 0; ni < fortran_vars.contractions(i); ni++) {
        for (nj = 0; nj < fortran_vars.contractions(j); nj++) {
          ai = fortran_vars.a_values(i,ni);
          aj = fortran_vars.a_values(j,nj);
          zeta = ai + aj;
          ksi = ai * aj / zeta;
          total_num_terms++;
          // TODO: right now, we're saving function values / nuclei # / density element for each thread; is there a better way to provide these values
          // to the kernel? Might be able to just send all function values/density matrix to the device and give each thread an index into the global arrays
          // Memory access patterns whon't be great, but they only get read in once
          if (dsq*ksi < fortran_vars.rmax) {
            //local2func1.push_back(i);
            a_values1.push_back(ai);
            //local2func2.push_back(j);
            a_values2.push_back(aj);
            num_terms++;
            cc_values.push_back(fortran_vars.c_values(i,ni)*fortran_vars.c_values(j,nj));
            nuclei1.push_back(nuc_i); nuclei2.push_back(nuc_j);
            dens_values.push_back(fortran_vars.rmm_input_ndens1.data[dens_ind]);
          }
        }
      }
      j += j_orbitals;
    }
    i += i_orbitals;
  }
  std::cout << "Number of significant Gaussian pairs: " << num_terms << std::endl;
  std::cout << "Total Gaussian pairs: " << total_num_terms << std::endl;

  // Pad the input so that out-of-range threads do a dummy calculation (same as the first thread), rather than branching and idling
  for (i = 0; i < QMMM_FORCES_BLOCK_SIZE - num_terms % QMMM_FORCES_BLOCK_SIZE; i++) {
    a_values1.push_back(a_values1[0]);
    a_values2.push_back(a_values2[0]);
    cc_values.push_back(cc_values[0]);
    dens_values.push_back(dens_values[0]);
    //local2func1.push_back(local2func1[0]);
    //local2func2.push_back(local2func2[0]);
    nuclei1.push_back(nuclei1[0]);
    nuclei2.push_back(nuclei2[0]);
  }
  // Send forces input to device (a values, thread function #s, thread nuclei #s)
  CudaMatrix<scalar_type> dev_a_values1(a_values1), dev_a_values2(a_values2), dev_cc_values(cc_values), dev_dens_values(dens_values);
  CudaMatrixUInt /*dev_func1(local2func1), dev_func2(local2func2),*/ dev_nuclei1(nuclei1), dev_nuclei2(nuclei2);

  //hipBindTextureToArray(qmmm_F_values_tex,gammaArray);

  /*dim3 testThreads(100,6);
  dim3 testBlock(32,6);
  dim3 testGrid(divUp(testThreads,testBlock));
  gpu_test_fmu_tex<scalar_type><<<testGrid,testBlock>>>( 0.5,GAMMA_INC );*/

  // Allocate output arrays on device (forces)
  CudaMatrix<vec_type<scalar_type,3> > gpu_partial_mm_forces, gpu_partial_qm_forces;//, gpu_mm_forces, gpu_qm_forces;

  gpu_partial_mm_forces.resize(COALESCED_DIMENSION(divUp(num_terms,QMMM_FORCES_BLOCK_SIZE)), fortran_vars.clatoms);
  gpu_partial_qm_forces.resize(COALESCED_DIMENSION(divUp(num_terms,QMMM_FORCES_BLOCK_SIZE)), fortran_vars.atoms);
  //gpu_mm_forces.resize(fortran_vars.clatoms,1);
  //gpu_qm_forces.resize(fortran_vars.atoms,1);

  dim3 threads(num_terms);
  dim3 blockSize(QMMM_FORCES_BLOCK_SIZE);
  dim3 gridSize = divUp(threads, blockSize);
  // Currently: density and c coefficents have 1-to-1 mapping to thread, and they only show up in the calculation multiplied together
  // So, if we were to keep this mapping, would just send the product
  // However, I'm leaving things as they are, sending the two individually, as this mapping probably isn't optimal:
  // -Density matrix maps 1-to-1 to a function x function set of terms (e.g., all terms (primitive i x primitive j) in a p_y x p_x set have the same density value)
  // -ci x cj coefficients map 1-to-1 to a primitive x primitive term (e.g., each term (primitive i x primitive j) in a p_y x p_x set have differenct c values, but the p_z x p_x term
  //                                                                       in the same sub-shell / sub-shell block has the same set of c values)
  // -ai x aj coefficients have the same mapping as ci x cj
  gpu_qmmm_forces<scalar_type><<<gridSize,blockSize>>>( num_terms, dev_a_values1.data, dev_a_values2.data, dev_cc_values.data,
                                                          dev_dens_values.data, /*dev_func1.data, dev_func2.data,*/ dev_nuclei1.data, dev_nuclei2.data,
                                                          gpu_partial_mm_forces.data, gpu_partial_qm_forces.data );//, fortran_vars.s_funcs, fortran_vars.s_funcs+fortran_vars.p_funcs*3 );

  HostMatrix<vec_type<scalar_type,3> > cpu_partial_mm_forces(gpu_partial_mm_forces), cpu_partial_qm_forces(gpu_partial_qm_forces);//cpu_mm_forces(gpu_mm_forces), cpu_qm_forces(gpu_qm_forces);

  // TODO: need to think about how to accumulate individual force terms
  // Currently, we reduce on a per-block basis in the kernel, then accumulate the block results here on the host
  // Maybe we could skip the reduction in the kernel (will speed it up, but by how much?), and each thread writes its own term to global memory, followed by a second kernel
  // that reduces all the individual thread terms (this is basically how the XC code works)
  // However, not sure that the memory requirements of each thread saving its term will be OK
  // Alternative: keep the kernel reduction, and reduce the block results in another kernel (rather than here on the host)
  for (i = 0; i < fortran_vars.atoms; i++) {
    qm_forces[i + 0 * fortran_vars.atoms] = 0;//cpu_qm_forces(i,0).x;
    qm_forces[i + 1 * fortran_vars.atoms] = 0;//cpu_qm_forces(i,0).y;
    qm_forces[i + 2 * fortran_vars.atoms] = 0;//cpu_qm_forces(i,0).z;
    for (j = 0; j < gridSize.x; j++) {
      qm_forces[i + 0 * fortran_vars.atoms] += cpu_partial_qm_forces(j,i).x;
      qm_forces[i + 1 * fortran_vars.atoms] += cpu_partial_qm_forces(j,i).y;
      qm_forces[i + 2 * fortran_vars.atoms] += cpu_partial_qm_forces(j,i).z;
    }
  }
  for (i = 0; i < fortran_vars.clatoms; i++) {
    mm_forces[i + 0 * (fortran_vars.atoms+fortran_vars.clatoms)] = 0;//cpu_mm_forces(i,0).x;
    mm_forces[i + 1 * (fortran_vars.atoms+fortran_vars.clatoms)] = 0;//cpu_mm_forces(i,0).y;
    mm_forces[i + 2 * (fortran_vars.atoms+fortran_vars.clatoms)] = 0;//cpu_mm_forces(i,0).z;
    for (j = 0; j < gridSize.x; j++) {
      mm_forces[i + 0 * (fortran_vars.atoms+fortran_vars.clatoms)] += cpu_partial_mm_forces(j,i).x;
      mm_forces[i + 1 * (fortran_vars.atoms+fortran_vars.clatoms)] += cpu_partial_mm_forces(j,i).y;
      mm_forces[i + 2 * (fortran_vars.atoms+fortran_vars.clatoms)] += cpu_partial_mm_forces(j,i).z;
    }
  }

  //hipUnbindTexture(qmmm_F_values_tex);

  cudaAssertNoError("qmmm");
}

template<class scalar_type>
void clean_gamma( void ) {
  scalar_type* d_str_ptr;
  hipMemcpyFromSymbol(&d_str_ptr,HIP_SYMBOL(gpu_str),sizeof(d_str_ptr));
  hipFree(d_str_ptr);
  //hipFreeArray(gammaArray);

  cudaAssertNoError("clean_gamma");
}
#if FULL_DOUBLE
template void get_qmmm_forces<double>(double* qm_forces, double* mm_forces);
template void clean_gamma<double>( void );
#else
template void get_qmmm_forces<float>(double* qm_forces, double* mm_forces);
template void clean_gamma<float>( void );
#endif

}
